#include "hip/hip_runtime.h"
/*! @file
  @brief
  Guru bytecode executor.

  <pre>
  Copyright (C) 2015-2018 Kyushu Institute of Technology.
  Copyright (C) 2015-2018 Shimane IT Open-Innovation Center.

  This file is distributed under BSD 3-Clause License.

  Fetch mruby VM bytecodes, decode and execute.

  </pre>
*/
#include <stdio.h>

#include "alloc.h"
#include "static.h"

#include "opcode.h"
#include "load.h"
#include "vm.h"

//================================================================
/*!@brief
  VM initializer.

  @param  vm  Pointer to VM
*/
__GURU__
void _mrbc_vm_begin(mrbc_vm *vm)
{

    MEMSET((uint8_t *)vm->regfile, 0, sizeof(vm->regfile));	// clean up registers

    vm->regfile[0].tt  	= MRBC_TT_CLASS;		// regfile[0] is self
    vm->regfile[0].cls 	= mrbc_class_object;	// root class

    vm->calltop = NULL;							// no call

    vm->pc 		= 0;							// starting IP
    vm->klass 	= mrbc_class_object;			// target class
    vm->reg 	= vm->regfile;					// pointer to reg[0]
    vm->pc_irep = vm->irep;						// root of irep tree
    vm->run   	= 1;
}

//================================================================
/*!@brief
  VM finalizer.

  @param  vm  Pointer to VM
*/
__GURU__ __INLINE__
void _mrbc_vm_end(mrbc_vm *vm)
{
#ifndef MRBC_DEBUG
    mrbc_free_all();
#endif
}

//================================================================
/*!@brief
  Fetch a bytecode and execute

  @param  vm    A pointer of VM.
  @retval 0  No error.
*/
__GURU__ __INLINE__
int _mrbc_vm_exec(mrbc_vm *vm)
{
	return mrbc_op(vm);
}

//================================================================
/*!@brief
  release mrbc_irep holds memory
*/
__GURU__
void _mrbc_free_irep(mrbc_irep *irep)
{
    // release pool.
    for(int i = 0; i < irep->plen; i++) {
        mrbc_free(irep->pool[i]);
    }
    if (irep->plen) mrbc_free(irep->pool);

    // release all child ireps.
    for(int i = 0; i < irep->rlen; i++) {
        _mrbc_free_irep(irep->irep_list[i]);
    }
    if (irep->rlen) mrbc_free(irep->irep_list);

    mrbc_free(irep);
}

__global__
void _run_vm(mrbc_vm *vm)
{
	if (threadIdx.x!=0 || blockIdx.x!=0) return;

	_mrbc_vm_begin(vm);
	_mrbc_vm_exec(vm);
	_mrbc_vm_end(vm);

	__syncthreads();
}

int guru_vm_init(guru_ses *ses)
{
	mrbc_vm *vm = (mrbc_vm *)guru_malloc(sizeof(mrbc_vm), 1);
	if (!vm) return -4;

	guru_parse_bytecode<<<1,1>>>(vm, ses->req);		// can also be done on host?
	hipDeviceSynchronize();

#ifdef MRBC_DEBUG
	printf("guru bytecode loaded:\n");
	dump_irep(vm->irep);
#endif
	ses->vm = (uint8_t *)vm;
	return 0;
}

int guru_vm_run(guru_ses *ses)
{
	int sz;
	hipDeviceGetLimit((size_t *)&sz, hipLimitStackSize);
	printf("defaultStackSize %d =>", sz);

	hipDeviceSetLimit(hipLimitStackSize, (size_t)sz*4);
	hipDeviceGetLimit((size_t *)&sz, hipLimitStackSize);
	printf("%d\n", sz);

	_run_vm<<<1,1>>>((mrbc_vm *)ses->vm);
	hipDeviceSynchronize();

	return 0;
}

#ifdef MRBC_DEBUG
void dump_irep(mrbc_irep *irep)
{
	printf("\tnregs=%d, nlocals=%d, pools=%d, syms=%d, reps=%d, ilen=%d\n",
			irep->nreg, irep->nlv, irep->plen, irep->slen, irep->rlen, irep->ilen);
	// dump all children ireps
	for (int i=0; i<irep->rlen; i++) {
		dump_irep(irep->irep_list[i]);
	}
}
#endif

