#include "hip/hip_runtime.h"
#include <stdio.h>
#include "c_ext.h"

#define SZ 512

__global__
void k_saxpy(int N, float a, float *x, float *y) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;	// blockDim.x = number of threads/block
	if (i < N) y[i] = a*x[i] + y[i];
}

__global__
void k_minit(int N, float *d_x, float *d_y) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;	// for (..., i+=n_threads) ...
	if (i < N) {									// while (i<N) ...
		d_x[i] = 1.0f;
	    d_y[i] = 2.0f;
	}
}

__device__ void d_sum(int t, float *sum) {
    for (int s=SZ; s>0; s>>=1) {
       if (t < s) sum[t] += sum[t + s]; 	// add second half of the block to the first half
       __syncthreads();						// dataflow flood gate
    }
}

__global__ void k_sum(int N, float *d_y) {	// sequentially executed in blocks per SM
    __shared__ float sum[2 * SZ];			// statically allocated on device

    int t = threadIdx.x;					// [0..511]
    int	i = blockIdx.x*2*SZ + t;			// global index (every 2 blocks)

    //@@ Load global array into shared memory
    sum[t]    = (i < N)    ? abs(d_y[i])-4.0    : 0.0;	// copy first half
    sum[SZ+t] = (SZ+i < N) ? abs(d_y[SZ+i])-4.0 : 0.0;	// copy second half

    d_sum(t, sum);							// call device function to reduce array

    //@@ Write the computed sum of the block to the block head
    if (t==0) d_y[blockIdx.x] = sum[0];
}

__global__ void k_sum2(int N, float *d_y) {	// sum front and back of entire array
	extern __shared__ float sum[];

	int t = threadIdx.x;					// in-block thread id
	int i = blockIdx.x*blockDim.x + t;		// in-grid global array index
	sum[t]= abs(d_y[i]-4.0f) +
			abs(d_y[i+blockDim.x]-4.0f);	// sum from 2 blocks to cut thread count in half

	for (int s=blockDim.x>>1; s>32; s>>=1) {// binary step reducing stride width
		if (t < s) sum[t] += sum[t + s];  	// sum [t] and [t+s] into [t] i.e. stride head
		__syncthreads();
	}
	if (t<32) {								// use unrolling to speed up 2x here
		sum[t]+=sum[t+32]; sum[t]+=sum[t+16]; sum[t]+=sum[t+8];
		sum[t]+=sum[t+4];  sum[t]+=sum[t+2];  sum[t]+=sum[t+1];
	}
	if (t==0) d_y[blockIdx.x] = sum[0]; 	// write back to each global block head
}

void echeck(const char *str) {
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) printf("%s, GPU OK\n", str);
    else {
    	printf("%s, GPU failed: %s\n", str, hipGetErrorString(err));
    	exit(-1);
    }
}

void edump(int N, float msec, float *y, float *d_y) {
	k_sum<<<(N+SZ-1)/SZ, SZ>>>(N, d_y);
	echeck("edump");

	hipMemcpy(y, d_y, sizeof(float)*(N+SZ-1)/SZ, hipMemcpyDeviceToHost);
//	hipDeviceSynchronize();

	float dif = 0.0;
	for (int i=0; i<(N+SZ-1)/SZ; i++)
		dif += y[i];
	printf("Max delta: %f, (Bandwidth %f GB/s, %f GFLOPs)\n", dif, N*4*3*1e-6/msec, N*2*1e-6/msec);
}

int do_cuda(void) {
  int N = 1<<24;

  float *x,   *y;
  float *d_x, *d_y;
  float *m_x, *m_y;

  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));
  hipMalloc(&d_x, N*sizeof(float));
  hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Perform SAXPY on 16M on-device elements
  hipEventRecord(start);
  k_saxpy<<<(N+SZ-1)/SZ, SZ>>>(N, 2.0f, d_x, d_y);               // 32K blocks
  hipEventRecord(stop);

  hipEventSynchronize(stop);
  echeck("H2D, saxpy, D2H");

  float msec = 0;
  hipEventElapsedTime(&msec, start, stop);
  edump(N, msec, y, d_y);

  hipMallocManaged(&m_x, N*sizeof(float));
  hipMallocManaged(&m_y, N*sizeof(float));

  k_minit<<<(N+SZ-1)/SZ, SZ>>>(N, m_x, m_y);
  hipDeviceSynchronize();
  echeck("managed mem init");

  // Perform SAXPY on 16M managed elements
  hipEventRecord(start);
  k_saxpy<<<(N+SZ-1)/SZ, SZ>>>(N, 2.0f, m_x, m_y);               // run on host?
  echeck("managed saxpy");
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&msec, start, stop);
  edump(N, msec, y, m_y);

  hipFree(m_x);
  hipFree(m_y);
  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);

  return 0;
}
