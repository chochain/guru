
#include <hip/hip_runtime.h>
#include <stdio.h>

#define SZ 512

__global__
void k_saxpy(int N, float a, float *x, float *y) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;	// blockDim.x = number of threads/block
	if (i < N) y[i] += a*x[i];						// C = aX+B in global memory
}

__global__
void k_minit(int N, float *d_x, float *d_y) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;	// for (..., i+=n_threads) ...
	if (i < N) {									// while (i<N) ...
		d_x[i] = 1.0f;
	    d_y[i] = 2.0f;
	}
}

__global__ void k_sum(int N, float *d_y) {	// sum front and back of entire array, destructive
    __shared__ float sum[SZ];				// statically allocated on device

	int t = threadIdx.x;					// in-block thread id
	int i = blockIdx.x*blockDim.x + t;		// in-grid global array index

	sum[t] = (i<N) ? d_y[i] : 0.0;			// copy the global value into

	for (int s=blockDim.x>>1; s>0; s>>=1) { // binary step reducing stride width
		if (t < s) sum[t] += sum[t + s];  	// sum [t] and [t+s] into [t] i.e. stride head
		__syncthreads();
	}
	if (t==0) d_y[blockIdx.x] = sum[0]; 	// write back to each global block head
}

__forceinline__ __device__ void d_sum2(int t, float *sum) {	// reduce array sum[2*SZ] into sum[0]
    for (int s=SZ; s>32; s>>=1) {			// folding by half the stride-size
       if (t < s) sum[t] += sum[t + s]; 	// add second half of the block to the first half
       __syncthreads();						// dataflow flood gate between warps
    }
    if (t<32) {								// unroll last warp, ~= 15% faster
    	sum[t]+=sum[t+32]; sum[t]+=sum[t+16]; sum[t]+=sum[t+8];
    	sum[t]+=sum[t+4];  sum[t]+=sum[t+2];  sum[t]+=sum[t+1];
    }
}

__global__ void k_sum_rec(int N, float *o_y, float *i_y) {	// recursively in blocks per SM
    __shared__ float sum[2 * SZ];			// hold double the thread count

    int t = threadIdx.x;					// [0..511]
    int	i = blockIdx.x*2*SZ + t;			// global index (every 2 blocks)

    sum[t]    = (i < N)    ? i_y[i]    : 0.0;	// copy first half into shared memory
    sum[SZ+t] = (SZ+i < N) ? i_y[SZ+i] : 0.0;	// copy second half

    d_sum2(t, sum);							// call device function to reduce array

    if (t==0) o_y[blockIdx.x] = sum[0];		// put sum into block head
}

__global__ void k_sum2(int N, float *o_y, float *i_y) {	// recursively in blocks per SM
    __shared__ float sum[2 * SZ];			// hold double the thread count

    int t = threadIdx.x;					// [0..511]
    int	i = blockIdx.x*2*SZ + t;			// global index (every 2 blocks)

    if (blockIdx.x==0) o_y[0] = 0.0;

    sum[t]    = (i < N)    ? i_y[i]    : 0.0;	// copy first half into shared memory
    sum[SZ+t] = (SZ+i < N) ? i_y[SZ+i] : 0.0;	// copy second half

    d_sum2(t, sum);							// call device function to reduce array

    if (t==0) o_y[0] += sum[0];				// put sum into block head
}

__global__ void k_sum_fast(int N, float *o_y, float *i_y) {	// sum front and back of entire array, destructive
    __shared__ float sum[SZ*2];				// hold double the thread count

	int t = threadIdx.x;					// thread id [0..511]
	int i = blockIdx.x*SZ*2 + t;			// in-block index 1024*[0..3] + [0..511]
	int B = gridDim.x*SZ*2;					// grid stride size

	sum[t]    = 0.0;
	sum[SZ+t] = 0.0;

	do {
		sum[t] 	  += (i < N)    ? i_y[i]    : 0.0;
		sum[SZ+t] += (SZ+i < N) ? i_y[SZ+i] : 0.0;
		i += B;								// advance one stride
	} while (i<=N);

	d_sum2(t, sum);

	if (t==0) o_y[blockIdx.x] = sum[0];		// put sum into block head
}

void echeck(const char *str) {
	hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) printf("\nOK> %s: ", str);
    else {
    	printf("\nERR> %s: %s\n", str, hipGetErrorString(err));
    	exit(-1);
    }
}

void bmark2(int N, float msec, float *d_y) {
	//	k_sum<<<(N+SZ-1)/SZ, SZ>>>(N, d_y);	// vanilla sum, SZ threads/block
	float *o_y;
	hipMalloc(&o_y, sizeof(float));				// allocate output array, sync here
	echeck("bmark2 malloc");

	float tot;
	k_sum2<<<(N+SZ*2-1)/SZ/2, SZ>>>(N, o_y, d_y);	// double-width blocks
	echeck("k_sum2()");
	hipMemcpy(&tot, o_y, sizeof(float), hipMemcpyDeviceToHost);	// warp sync here

	printf("\nTotal: %f, (Bandwidth %f GB/s, %f GFLOPs)\n", tot, N*4*3*1e-6/msec, N*2*1e-6/msec);
	hipFree(o_y);							// release, async
}

void bmark_rec(int N, float msec, float *d_y) {
	int n = N, SZ2 = SZ*2;
	int nblk = (n+SZ2-1)/SZ2;				// double-width block count

	float *o_y, *i_y = d_y;
	hipMalloc(&o_y, (nblk>1 ? nblk : 2)*sizeof(float));				// allocate output array, sync here
	echeck("bmark_rec malloc");

	float v[2];
	do {		// recursively down to 1 value
		k_sum_rec<<<nblk, SZ>>>(n, o_y, i_y);
		echeck("k_sum2()");
		hipMemcpy(&v, o_y, sizeof(float)*2, hipMemcpyDeviceToHost);	// warp sync here
		printf("n=%d, nblk=%d: v[0]=%f, v[1]=%f", n, nblk, v[0], v[1]);
		n    = nblk;
		nblk = (n+SZ2-1)/SZ2;				// reduce by 2*block count
		i_y  = o_y;							// point to output array
	} while (n>1);

	printf("\nTotal: %f, (Bandwidth %f GB/s, %f GFLOPs)\n", v[0], N*4*3*1e-6/msec, N*2*1e-6/msec);
	hipFree(o_y);							// release, async
}

void bmark_fast(int N, float msec, float *d_y) {
	int NBLK = 12;							// number_of_sm * max_threads_per_sm / threads_per_block

	float *o_y;
	hipMalloc(&o_y, NBLK*sizeof(float));	// allocate output array, sync here
	echeck("bmark_fast malloc");

	k_sum_fast<<<NBLK, SZ>>>(N, o_y, d_y);	// number_of_sm * max_threads_per_sm / threads_per_block

	float v[NBLK];
	double tot = 0.0;
	hipMemcpy(&v, o_y, NBLK*sizeof(float), hipMemcpyDeviceToHost);	// warp sync here

	for (int i=0; i<NBLK; i++) tot += v[i];	// hand back to CPU to tally up, takes less time

	printf("\nTotal: %f, (Bandwidth %f GB/s, %f GFLOPs)\n", tot, N*4*3*1e-6/msec, N*2*1e-6/msec);
	hipFree(o_y);							// release, async
}

int do_cuda(void) {
  int N = (1<<24);							// max digit of float precision

  float *x, *y, *d_x, *d_y, *m_x, *m_y;

  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));
  hipMalloc(&d_x, N*sizeof(float));
  hipMalloc(&d_y, N*sizeof(float));

  for (int i=0; i<N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  hipEvent_t ev0, ev1;
  hipEventCreate(&ev0);
  hipEventCreate(&ev1);

  // Perform SAXPY on 16M on-device elements
  hipEventRecord(ev0);
  k_saxpy<<<(N+SZ-1)/SZ, SZ>>>(N, 2.0f, d_x, d_y);               // 32K blocks
  hipEventRecord(ev1);
  echeck("H2D, saxpy, D2H");

  float msec = 0;
  hipEventElapsedTime(&msec, ev0, ev1);
  bmark_rec(N, msec, d_y);
  bmark2(N, msec, d_y);											// benchmark, external recursive sum

  hipMallocManaged(&m_x, N*sizeof(float));
  hipMallocManaged(&m_y, N*sizeof(float));

  k_minit<<<(N+SZ-1)/SZ, SZ>>>(N, m_x, m_y);
  echeck("managed mem init");

  // Perform SAXPY on 16M managed elements
  hipEventRecord(ev0);
  k_saxpy<<<(N+SZ-1)/SZ, SZ>>>(N, 2.0f, m_x, m_y);              // run on host?
  hipEventRecord(ev1);
  echeck("managed saxpy");
  hipEventElapsedTime(&msec, ev0, ev1);
  bmark_fast(N, msec, m_y);										// benchmark, internal loop sum

  hipEventDestroy(ev0);										// release event objects
  hipEventDestroy(ev1);
  hipFree(m_x);
  hipFree(m_y);

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);

  return 0;
}

int main(int argc, char **argv)
{
    do_cuda();
    return 0;
}


