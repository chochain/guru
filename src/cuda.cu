#include "hip/hip_runtime.h"
#include <stdio.h>
#include "c_ext.h"

__global__
void saxpy(int N, float a, float *x, float *y)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < N) y[i] = a*x[i] + y[i];
}

__global__
void d_m_init(int N, float *d_x, float *d_y)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < N) {
		d_x[i] = 1.0f;
	    d_y[i] = 2.0f;
	}
}

void edump(int N, float msec, float *y)
{
	float dif = 0.0f;
	for (int i = 0; i < N; i++)
		dif = max(dif, abs(y[i] - 4.0f));
	printf("Max delta: %f, (Bandwidth %f GB/s, %f GFLOPs)\n", dif, N*4*3*1e-6/msec, N*2*1e-6/msec);
}

int do_cuda(void)
{
  int N = 1<<24;
  float *x, *y, *d_x, *d_y;
  float *m_x, *m_y;

  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));
  hipMalloc(&d_x, N*sizeof(float));
  hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Perform SAXPY on 16M on-device elements
  hipEventRecord(start);
  saxpy<<<(N+511)/512, 512>>>(N, 2.0f, d_x, d_y);               // async
  hipEventRecord(stop);

  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);  // continue running
  hipEventSynchronize(stop);
  float msec = 0;
  hipEventElapsedTime(&msec, start, stop);
  edump(N, msec, y);

  hipMallocManaged(&m_x, N*sizeof(float));
  hipMallocManaged(&m_y, N*sizeof(float));

  d_m_init<<<(N+511)/512, 512>>>(N, m_x, m_y);

  // Perform SAXPY on 16M managed elements
  hipEventRecord(start);
  saxpy<<<(N+511)/512, 512>>>(N, 2.0f, m_x, m_y);               // run on host?
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&msec, start, stop);
  edump(N, msec, m_y);

  hipFree(m_x);
  hipFree(m_y);
  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);

  return 0;
}
