#include "hip/hip_runtime.h"
/*! @file
  @brief
  GURU value and macro definitions

  <pre>
  Copyright (C) 2019- GreenII

  This file is distributed under BSD 3-Clause License.
  </pre>
*/
#include "value.h"
#include "object.h"

#include "c_string.h"
#include "c_array.h"
#include "c_hash.h"
#include "c_range.h"

//================================================================
/*! compare
 */
__GURU__ S32
_string_cmp(const GV *v0, const GV *v1)
{
	S32 x  = (U32)v0->str->bsz - (U32)v1->str->bsz;
	if (x) return x;

	return STRCMP(v0->str->raw, v1->str->raw);
}

//================================================================
/*! compare two GVs

  @param  v1	Pointer to GV
  @param  v2	Pointer to another GV
  @retval 0	v1 == v2
  @retval plus	v1 >  v2
  @retval minus	v1 <  v2
*/
__GURU__ S32
guru_cmp(const GV *v0, const GV *v1)
{
    if (v0->gt != v1->gt) { 						// GT different
#if GURU_USE_FLOAT
    	GF f0, f1;

        if (v0->gt==GT_INT && v1->gt==GT_FLOAT) {
            f0 = v0->i;
            f1 = v1->f;
            return -1 + (f0 == f1) + (f0 > f1)*2;	// caution: NaN == NaN is false
        }
        if (v0->gt==GT_FLOAT && v1->gt==GT_INT) {
            f0 = v0->f;
            f1 = v1->i;
            return -1 + (f0 == f1) + (f0 > f1)*2;	// caution: NaN == NaN is false
        }
#endif // GURU_USE_FLOAT
        // leak Empty?
        if ((v0->gt==GT_EMPTY && v1->gt==GT_NIL) ||
            (v0->gt==GT_NIL   && v1->gt==GT_EMPTY)) return 0;

        // other case
        return v0->gt - v1->gt;
    }

    // check value
    switch(v1->gt) {
    case GT_NIL:
    case GT_FALSE:
    case GT_TRUE:   return 0;
    case GT_INT:
    case GT_SYM: 	return -1 + (v0->i==v1->i) + (v0->i > v1->i)*2;
#if GURU_USE_FLOAT
    case GT_FLOAT:  return -1 + (v0->f==v1->f) + (v0->f > v1->f)*2;	// caution: NaN == NaN is false
#endif // GURU_USE_FLOAT

    case GT_CLASS:
    case GT_OBJ:
    case GT_PROC:   return -1 + (v0->self==v1->self) + (v0->self > v1->self)*2;
    case GT_STR: 	return _string_cmp(v0, v1);
#if GURU_USE_ARRAY
    case GT_ARRAY:  return guru_array_cmp(v0, v1);
    case GT_RANGE:  return guru_range_cmp(v0, v1);
    case GT_HASH:   return guru_hash_cmp(v0, v1);
#endif // GURU_USE_ARRAY
    default:
        return 1;
    }
}

//================================================================
/*!@brief

  convert ASCII string to integer Guru version

  @param  s	source string.
  @param  base	n base.
  @return	result.
*/
__GURU__ GI
guru_atoi(const U8 *s, U32 base)
{
    GI  ret  = 0;
    U32 sign = 0;

REDO:
    switch(*s) {
    case '-': sign = 1;		// fall through.
    case '+': s++;	        break;
    case ' ': s++;          goto REDO;
    }

    U8  ch;
    U32 n;
    while ((ch = *s++) != '\0') {
        if      ('a' <= ch) 			 n = ch - 'a' + 10;
        else if ('A' <= ch) 			 n = ch - 'A' + 10;
        else if ('0' <= ch && ch <= '9') n = ch - '0';
        else break;

        if (n >= base) break;

        ret = ret * base + n;
    }
    return (sign) ? -ret : ret;
}

__GURU__ GF
guru_atof(const U8 *s)
{
#if GURU_USE_FLOAT
    int sign = 1, esign = 1, state=0;
    int r = 0, e = 0;
    long v = 0L, f = 0L;

    while ((*s<'0' || *s>'9') && *s!='+' && *s!='-') s++;

    if (*s=='+' || *s=='-') sign = *s++=='-' ? -1 : 1;

    while (*s!='\0' && *s!='\n' && *s!=' ' && *s!='\t') {
    	if      (state==0 && *s>='0' && *s<='9') {	// integer
    		v = (*s - '0') + v * 10;
    	}
    	else if (state==1 && *s>='0' && *s<='9') {	// decimal
    			f = (*s - '0') + f * 10;
    			r--;
        }
    	else if (state==2) {						// exponential
            if (*s=='-') {
                esign = -1;
                s++;
            }
            if (*s>='0' && *s<='9') e = (*s - '0') + e * 10;
        }
        state = (*s=='e' || *s=='E') ? 2 : ((*s=='.') ? 1 : state);
        s++;
    }
    GF ret = sign
    		* (v + (f==0 ? 0.0 : f * exp10((double)r)))
    		* (e==0 ? 1.0 : exp10((double)esign * e));

    return ret;
#else
    return 0.0;
#endif // GURU_USE_FLOAT
}

__GURU__ void
guru_memcpy(U8 *d, const U8 *s, U32 bsz)
{
    for (U32 i=0; s && d && i<bsz; i++, *d++ = *s++);
}

__GURU__ void
guru_memset(U8 *d, U8 v,  U32 bsz)
{
    for (U32 i=0; d && i<bsz; i++, *d++ = v);
}

__GURU__ int
guru_memcmp(const U8 *d, const U8 *s, U32 bsz)
{
	U32 i;
    for (i=0; i<bsz && *d==*s; i++, d++, s++);

    return i<bsz ? (*d - *s) : 0;
}

__GURU__ __INLINE__ void
_next_utf8(U8 **sp)
{
	U8  c = **sp;
	U32 b = 0;
	if      (c>0 && c<=127) 		b=1;
	else if ((c & 0xE0) == 0xC0) 	b=2;
	else if ((c & 0xF0) == 0xE0) 	b=3;
	else if ((c & 0xF8) == 0xF0) 	b=4;
	else *sp=NULL;					// invalid utf8

	*sp+=b;
}

__GURU__ U32
guru_strlen(const U8 *str, U32 use_byte)
{
	U32 n  = 0;
	U8  *s = (U8*)str;
	for (U32 i=0; s && *s!='\0'; i++, n++) {
		_next_utf8(&s);
	}
	return (s && use_byte) ? s - str : n;
}

__GURU__ U8 *
guru_strcut(const U8 *str, U32 n)
{
	U8 *s = (U8*)str;
	for (U32 i=0, c=0; n>0 && s && *s!='\0'; i++) {
		_next_utf8(&s);
		if (++c >= n) break;
	}
	return s;
}

__GURU__ void
guru_strcpy(U8 *d, const U8 *s)
{
    guru_memcpy(d, s, STRLENB(s)+1);
}

__GURU__ S32
guru_strcmp(const U8 *s1, const U8 *s2)
{
    return guru_memcmp(s1, s2, STRLENB(s1));
}

__GURU__ U8*
guru_strchr(U8 *s, const U8 c)
{
    while (s && *s!='\0' && *s!=c) s++;

    return (U8*)((*s==c) ? &s : NULL);
}

__GURU__ U8*
guru_strcat(U8 *d, const U8 *s)
{
	guru_memcpy(d+STRLENB(d), s, STRLENB(s)+1);
    return d;
}

__GURU__ GV NIL() 	{ GV v; { v.gt=GT_NIL;   v.acl=0; } return v; }
__GURU__ GV EMPTY()	{ GV v; { v.gt=GT_EMPTY; v.acl=0; } return v; }


