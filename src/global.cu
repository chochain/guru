#include "hip/hip_runtime.h"
#include <assert.h>
#include "value.h"
#include "global.h"
/*

  GLobal objects are stored in 'mrbc_global' array.
  'mrbc_global' array is decending order by sym_id.
  In case of searching a global object, binary search is used.
  In case of adding a global object, insertion sort is used.

*/
typedef enum {
    GURU_GLOBAL_OBJECT = 1,
    GURU_CONST_OBJECT,
} mrbc_gtype;

typedef struct mrbc_gobject_ {
    mrbc_gtype 	gtype 	:8;
    mrbc_sym 	sym_id;
    mrbc_object obj;
} mrbc_gobject;

// max of global object in mrbc_global[]
__GURU__ int _mutex_glb;
__GURU__ int _global_end;
__GURU__ mrbc_gobject _mrbc_global[MAX_GLOBAL_OBJECT_SIZE];

/* search */
/* linear search is not efficient! */
/* TODO: Use binary search */
__GURU__ int
_get_idx(mrbc_sym sid, mrbc_gtype gtype)
{
    for (int i=0 ; i<_global_end ; i++) {
        mrbc_gobject *obj = &_mrbc_global[i];
        if (obj->sym_id == sid && obj->gtype == gtype) return i;
    }
    return -1;
}

__GURU__ mrbc_value
_get_obj(mrbc_sym sid, mrbc_gtype gtype)
{
    int index = _get_idx(sid, gtype);
    if (index < 0) mrbc_nil_value();

    mrbc_retain(&_mrbc_global[index].obj);
    return _mrbc_global[index].obj;
}

/* add */
/* TODO: Check reference count */
__GURU__ void
global_object_add(mrbc_sym sid, mrbc_value v)
{
    int idx = _get_idx(sid, GURU_GLOBAL_OBJECT);

    MUTEX_LOCK(_mutex_glb);

    if (idx == -1) {
        idx = _global_end++;
        assert(idx < MAX_GLOBAL_OBJECT_SIZE);	// maybe raise ex
    }
    else {
        mrbc_release(&(_mrbc_global[idx].obj));
    }
    _mrbc_global[idx].gtype  = GURU_GLOBAL_OBJECT;
    _mrbc_global[idx].sym_id = sid;
    _mrbc_global[idx].obj    = v;

    MUTEX_FREE(_mutex_glb);
    
    mrbc_retain(&v);
}

__GURU__ void
const_object_add(mrbc_sym sid, mrbc_object *obj)
{
    int idx = _get_idx(sid, GURU_CONST_OBJECT);

    MUTEX_LOCK(_mutex_glb);

    if (idx == -1) {
        idx = _global_end++;
        assert(idx < MAX_GLOBAL_OBJECT_SIZE);	// maybe raise ex
    }
    else {
        // warning: already initialized constant.
        mrbc_release(&(_mrbc_global[idx].obj));
    }
    _mrbc_global[idx].gtype  = GURU_CONST_OBJECT;
    _mrbc_global[idx].sym_id = sid;
    _mrbc_global[idx].obj    = *obj;

    MUTEX_FREE(_mutex_glb);

    mrbc_retain(obj);
}

/* get */
__GURU__ mrbc_value
global_object_get(mrbc_sym sid)
{
    return _get_obj(sid, GURU_GLOBAL_OBJECT);
}

/* add const */
__GURU__ mrbc_object
const_object_get(mrbc_sym sid)
{
    return _get_obj(sid, GURU_CONST_OBJECT);
}
//
__GPU__ void
guru_global_init(void)
{
	if (blockIdx.x!=0 || threadIdx.x!=0) return;

	_mutex_glb  = 0;
	_global_end = 0;
}




