#include "hip/hip_runtime.h"
/*! @file
  @brief
  GURU 32-bit memory management.

  <pre>
  Copyright (C) 2019 GreenII.

  This file is distributed under BSD 3-Clause License.

  Memory management for objects in GURU.

  </pre>
*/
#include <stdio.h>
#include <assert.h>
#include "alloc.h"
#include "alloc32.h"

// TLSF: Two-Level Segregated Fit allocator with O(1) time complexity.
// Layer 1st(f), 2nd(s) model, smallest block 16-bytes, 16-byte alignment
// TODO: multiple-pool, thread-safe
// semaphore
__GURU__ volatile U32 	_mutex_mem;

// memory pool
__GURU__ U8				*_memory_pool;

// free memory bitmap
__GURU__ U32 			_l1_map;								// use lower 24 bits
__GURU__ U16 			_l2_map[L1_BITS];						// use all 16 bits
__GURU__ free_block		*_free_list[FL_SLOTS];

//================================================================
// most significant bit that is set
__GURU__ __INLINE__ U32
__fls(U32 x)
{
	U32 n;
	asm("bfind.u32 %0, %1;\n\t" : "=r"(n) : "r"(x));
	return n;
}

// least significant bit that is set
__GURU__ __INLINE__ U32
__ffs(U32 x)
{
	U32 n;
	asm(
		"brev.b32 %0, %1;\n\t"
		"clz.b32 %0, %0;\n\t"
		: "=r"(n) : "r"(x)
	);
	return n;
}
//================================================================
/*! calc f and s, and returns fli,sli of free_blocks

  @param  alloc_size	alloc size
  @retval int		index of free_blocks
*/
__GURU__ U32
__idx(U32 sz, U32P l1, U32P l2)
{
	U32 v = __fls(sz);
	U32 x = __ffs(sz);

    *l1 = v<BASE_BITS ? 0 : v - BASE_BITS;			// 1st level index
    *l2 = (sz >> (v - MN_BITS)) & L2_MASK;  // 2nd level index (with lower bits)

    return INDEX(*l1, *l2);
}

//================================================================
/*! wipe the free_block from linked list

  @param  blk	pointer to free block.
*/
__GURU__ void
__unmap(free_block *blk)
{
	assert(IS_FREE(blk));					// ensure block is free

    if (blk->prev) {						// down link exists
    	// blk->prev->next = blk->next;
    	PREV_FREE(blk)->next = U8POFF(NEXT_FREE(blk), PREV_FREE(blk));
    }
    else {			// top of the link, clear the map first (i.e. make available)
        U32 l1, l2;
        U32 index = __idx(blk->bsz, &l1, &l2);

        if ((_free_list[index]=NEXT_FREE(blk))==NULL) {
        	CLEAR_MAP(index);				// mark as unallocated
        }
    }
    if (blk->next) {						// up link
    	// blk->next->prev = blk->prev;
    	NEXT_FREE(blk)->prev = U8POFF(PREV_FREE(blk), NEXT_FREE(blk));
    }
    blk->next = blk->prev = 0;				// wipe for debugging
}

//================================================================
/*! merge p0 and p1 adjacent free blocks.
  ptr2 will disappear

  @param  ptr1	pointer to free block 1
  @param  ptr2	pointer to free block 2
*/
__GURU__ void
__pack(free_block *b0, free_block *b1)
{
	assert((free_block*)BLK_AFTER(b0)==b1);
	assert(IS_FREE(b1));

	// remove b0, b1 from free list first (sizes will not change)
    __unmap(b1);

	// merge p0 and p1
	used_block *b2 = (used_block *)BLK_AFTER(b1);
	b2->psz += b1->psz & ~FREE_FLAG;	// watch for the block->flag
    b0->bsz += b1->bsz;					// include the block header

#if GURU_DEBUG
    *((U64*)b1) = 0xeeeeeeeeeeeeeeee;	// wipe b1 header
#endif
}

//================================================================
/*! Mark that block free and register it in the free index table.

  @param  blk	Pointer to block to be freed.

  TODO: check thread safety
*/
__GURU__ void
_mark_free(free_block *blk)
{
	assert(IS_USED(blk));

	U32 l1, l2;
	U32 index = __idx(blk->bsz, &l1, &l2);

    U32 l1x= L1(index);
    U32 l2x= L2(index);
    U32 t1 = TIC(l1x);
    U32 t2 = TIC(l2x);
    U32 m1 = L1_MAP(index);
    U16 m2 = L2_MAP(index);

    SET_MAP(index);										// set free block available ticks

    U32 m1x = L1_MAP(index);
    U16 m2x = L2_MAP(index);

    // update block attributes
    free_block *head = _free_list[index];

    assert(head != blk);

    SET_FREE(blk);
    blk->next = head ? U8POFF(head, blk) : 0;			// setup linked list
    blk->prev = 0;
    if (head) {											// non-end block, add backward link
    	head->prev = U8POFF(blk, head);
    }
    _free_list[index] = blk;							// new head of the linked list
}

__GURU__ free_block*
_mark_used(U32 index)
{
    free_block *blk  = _free_list[index];
    assert(blk);
    assert(IS_FREE(blk));

    free_block *next = NEXT_FREE(blk);
    if (next) {											// next free block exists
        free_block *prev = PREV_FREE(blk);
    	next->prev = prev ? U8POFF(prev, next) : 0;		// up link
        _free_list[index] = next;						// take it out of free list
    }
    else {
        U32 l1x= L1(index);
        U32 l2x= L2(index);
        U32 t1 = TIC(l1x);
        U32 t2 = TIC(l2x);
        U32 m1 = L1_MAP(index);
        U16 m2 = L2_MAP(index);

        CLEAR_MAP(index);						// release the index

        U32 m1x = L1_MAP(index);
        U16 m2x = L2_MAP(index);

        if (L1_MAP(index)==0 && L2_MAP(index)==0) {
        	_free_list[index] = NULL;
        }
    }
    SET_USED(blk);

    return blk;
}

__GURU__ void
_merge_with_next(free_block *b1)
{
	free_block *b2 = (free_block *)BLK_AFTER(b1);
	while (b2 && IS_FREE(b2)) {
		__pack(b1, b2);
		b2 = (free_block *)BLK_AFTER(b1);	// try the already expanded block again
	}
}

__GURU__ free_block*
_merge_with_prev(free_block *b1)
{
    free_block *b0 = (free_block *)BLK_BEFORE(b1);
	if (IS_USED(b0)) return b1;

	__unmap(b0);							// take it out of free_list before merge
	__pack(b0, b1);							// take b1 out and merge with b0

	SET_USED(b0);							// _mark_free assume b0 to be a USED block
	_mark_free(b0);

    return b0;
}

//================================================================
/*! Find index to a free block

  @param  size	size
  @retval -1	not found
  @retval index to available _free_list
*/
__GURU__ S32
_find_free_index(U32 sz)
{
	U32 l1, l2;
    U32 index = __idx(sz, &l1, &l2);	// find free_list index by size

    if (_free_list[index]) return index;		// free block available, use it

    // no previous block exist, create a new one
    U32 avl = _l2_map[l1];			    // check any 2nd level available
    if (avl) {
    	l2 = __fls(avl);				// get first available l2 index
    }
    else if ((avl = _l1_map)) {			// check if 1st level available
        l1 = __fls(avl);        		// allocate new 1st & 2nd level indices
        l2 = __fls(_l2_map[l1]);
    }
    else return -1;						// out of memory
    return INDEX(l1, l2);               // index to freelist head
}

//================================================================
/*! Split free block by size (before allocating)

  @param  blk	pointer to free block
  @param  size	storage size
*/
__GURU__ void
_split(free_block *blk, U32 bsz)
{
	assert(IS_USED(blk));

    if ((bsz + MIN_BLOCK) > blk->bsz) return;	 				// too small to split 											// too small to split

    // split block, free
    free_block *free = (free_block *)U8PADD(blk, bsz);			// future next block (i.e. alot bsz bytes)
    free_block *aft  = (free_block *)BLK_AFTER(blk);			// next adjacent block

    free->bsz = blk->bsz - bsz;									// carve out the acquired block
    free->psz = U8POFF(free, blk);								// positive offset to previous block

    if (aft) {
        aft->psz = U8POFF(aft, free);							// backward offset (positive)
        _merge_with_next(free);									// _combine if possible
    }
    _mark_free(free);			// add to free_list and set (free, tail, next, prev) fields

    blk->bsz  = bsz;											// reduce size
}

//================================================================
/*! initialize

  @param  ptr	pointer to free memory block.
  @param  size	size. (max 4G)
*/
__GURU__ void
_init_mmu(void *mem, U32 size)
{
    assert(size > 0);

    U32 bsz = size - sizeof(free_block);

    _mutex_mem	 = 0;
    _memory_pool = (U8P)mem;

    // initialize entire memory pool as the first block
    free_block *blk  = (free_block *)_memory_pool;
    blk->bsz = bsz;						// 1st (big) block
    blk->psz = 0;
    SET_USED(blk);

    _mark_free(blk);					// will set free, tail, next, prev

    blk = (free_block *)BLK_AFTER(blk);	// last block
    blk->bsz = blk->next = blk->prev = 0;
    blk->psz = bsz;
    SET_USED(blk);
}

//================================================================
/*! allocate memory

  @param  size	request storage size.
  @return void* pointer to a guru memory block.
*/
__GURU__ void*
guru_alloc(U32 sz)
{
    U32 bsz = sz + sizeof(used_block);			// logical => physical size

    CHECK_ALIGN(bsz);							// assume caller already align the size
    CHECK_MINSZ(bsz);							// check minimum allocation size

	MUTEX_LOCK(_mutex_mem);

	U32 index 		= _find_free_index(bsz);
	free_block *blk = _mark_used(index);		// take the indexed block off free list

	_split(blk, bsz);							// allocate the block, free up the rest
#if GURU_DEBUG
    U32P p = (U32P)BLK_DATA(blk);				// point to raw space allocated
    for (U32 i=0; i < sz>>2; i++) *p++ = 0xaaaaaaaa;
#endif
	MUTEX_FREE(_mutex_mem);

	return BLK_DATA(blk);						// pointer to raw space
}

//================================================================
/*! re-allocate memory

  @param  ptr	Return value of raw malloc()
  @param  size	request size
  @return void* pointer to allocated memory.
*/
__GURU__ void*
guru_realloc(void *p0, U32 sz)
{
	U32 bsz = sz + sizeof(used_block);						// include the header

	CHECK_NULL(p0);
	CHECK_ALIGN(bsz);

    used_block *blk = (used_block *)BLK_HEAD(p0);
    assert(IS_USED(blk));									// make sure it is used

    if (bsz > blk->bsz) {
    	_merge_with_next((free_block *)blk);				// try to get the block bigger
    }
    if (bsz == blk->bsz) return p0;							// same size, good fit
    if (bsz < blk->bsz) {									// a little to big, split if we can
        _split((free_block *)blk, bsz);						// keep only the first bsz bytes
        return p0;
    }

    // not big enough block found, new alloc and deep copy
    void *p1 = guru_alloc(sz);
    memcpy(p1, p0, sz);										// deep copy

    guru_free(p0);											// reclaim block

    return p1;
}

//================================================================
/*! release memory
*/
__GURU__ void
guru_free(void *ptr)
{
	MUTEX_LOCK(_mutex_mem);

    free_block *blk = (free_block *)BLK_HEAD(ptr);			// get block header

    _merge_with_next(blk);
    _mark_free(blk);

    // the block is free now, try to merge a free block before if exists
    blk = _merge_with_prev(blk);

#if GURU_DEBUG
    if (BLK_AFTER(blk)) {
    	U32 *p = (U32*)U8PADD(blk, sizeof(free_block));
    	U32 sz = (blk->bsz - sizeof(free_block))>>2;
    	for (U32 i=0; i< (sz>32 ? 32 : sz); i++) *p++=0xffffffff;
    }
#endif

    MUTEX_FREE(_mutex_mem);
}

//================================================================
/*! release memory, vm used.

  @param  vm	pointer to VM.
*/
__GURU__ void
guru_memory_clr()
{
    used_block *p = (used_block *)_memory_pool;
    while (p) {
    	if (IS_USED(p)) {
    		guru_free(BLK_DATA(p));		// pointer to raw space
    	}
    	p = (used_block *)BLK_AFTER(p);
    }
}

#if GURU_DEBUG
//================================================================
/*! statistics

  @param  *total	returns total memory.
  @param  *used		returns used memory.
  @param  *free		returns free memory.
  @param  *fragment	returns memory fragmentation
*/
__GPU__ void
_alloc_stat(U32 v[])
{
	if (threadIdx.x!=0 || blockIdx.x!=0) return;

	U32 total=0, nfree=0, free=0, nused=0, used=0, nblk=0, nfrag=0;

	used_block *p = (used_block *)_memory_pool;

	U32 flag = IS_FREE(p);				// starting block type
	while (p) {	// walk the memory pool
		if (flag != IS_FREE(p)) {       // supposed to be merged
			nfrag++;
			flag = IS_FREE(p);
		}
		total += p->bsz;
		nblk  += 1;
		if (IS_FREE(p)) {
			nfree += 1;
			free  += p->bsz;
		}
		else {
			nused += 1;
			used  += p->bsz;
		}
		p = (used_block *)BLK_AFTER(p);
	}
	v[0] = total + sizeof(free_block);
	v[1] = nfree;
	v[2] = free;
	v[3] = nused;
	v[4] = used;
	v[5] = nblk;
	v[6] = nfrag;

	__syncthreads();
}

__GPU__ void
guru_memory_init(void *ptr, U32 sz)
{
	if (threadIdx.x!=0 || blockIdx.x!=0) return;

	_init_mmu(ptr, sz);
}

__HOST__ void*
cuda_malloc(U32 sz, U32 type)
{
	void *mem;

	switch (type) {
	case 0: 	hipMalloc(&mem, sz); break;			// allocate device memory
	default: 	hipMallocManaged(&mem, sz);			// managed (i.e. paged) memory
	}
    if (hipSuccess != hipGetLastError()) return NULL;

    return mem;
}

__HOST__ void
_get_alloc_stat(U32 stat[])
{
	U32P v;
	hipMallocManaged(&v, 8*sizeof(int));				// allocate host memory

	_alloc_stat<<<1,1>>>(v);
	hipDeviceSynchronize();

	for (U32 i=0; i<8; i++) {
		stat[i] = v[i];									// mirror stat back from device
	}
	hipFree(v);
}

__HOST__ void
guru_dump_alloc_stat(U32 trace)
{
	if (trace==0) return;

	U32 s[8];
	_get_alloc_stat(s);

	printf("\tmem=%d(0x%x): free=%d(0x%x), used=%d(0x%x), nblk=%d, nfrag=%d, %d%% allocated\n",
			s[0], s[0], s[1], s[2], s[3], s[4], s[5], s[6], (int)(100*(s[4]+1)/s[0]));
}
#endif
