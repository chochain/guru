#include "hip/hip_runtime.h"
/*! @file
  @brief
  mruby/c Array class

  <pre>
  Copyright (C) 2015-2018 Kyushu Institute of Technology.
  Copyright (C) 2015-2018 Shimane IT Open-Innovation Center.

  This file is distributed under BSD 3-Clause License.

  </pre>
*/

#include "vm_config.h"
#include <string.h>
#include <assert.h>

#include "guru.h"
#include "alloc.h"
#include "static.h"

#include "console.h"
#include "sprintf.h"

#include "opcode.h"
#include "class.h"

#include "object.h"
#include "c_array.h"
#include "c_string.h"

/*
  function summary

 (constructor)
    mrbc_array_new

 (destructor)
    mrbc_array_delete

 (setter)
  --[name]-------------[arg]---[ret]-------------------------------------------
    mrbc_array_set		*T		int
    mrbc_array_push		*T		int
    mrbc_array_unshift	*T		int
    mrbc_array_insert	*T		int

 (getter)
  --[name]-------------[arg]---[ret]---[note]----------------------------------
    mrbc_array_get		T		Data remains in the container
    mrbc_array_pop		T		Data does not remain in the container
    mrbc_array_shift	T		Data does not remain in the container
    mrbc_array_remove	T		Data does not remain in the container

 (others)
    mrbc_array_resize
    mrbc_array_clear
    mrbc_array_compare
    mrbc_array_minmax
*/

//================================================================
/*! get size
 */
__GURU__
int mrbc_array_size(const mrbc_value *ary)
{
    return ary->array->n;
}

//================================================================
/*! constructor

  @param  vm	pointer to VM.
  @param  size	initial size
  @return 	array object
*/
__GURU__
mrbc_value mrbc_array_new(int size)
{
    mrbc_value ret = {.tt = MRBC_TT_ARRAY};
    mrbc_array *h 	 = (mrbc_array *)mrbc_alloc(sizeof(mrbc_array));		// handle
    if (!h) return ret;	// ENOMEM

    mrbc_value *data = (mrbc_value *)mrbc_alloc(sizeof(mrbc_value) * size);	// buffer
    if (!data) {			// ENOMEM
        mrbc_free(h);
        return ret;
    }
    h->refc = 1;			// handle is referenced
    h->tt 	= MRBC_TT_ARRAY;
    h->size = size;
    h->n  	= 0;
    h->data = data;

    ret.array = h;

    return ret;
}

//================================================================
/*! destructor

  @param  ary	pointer to target value
*/
__GURU__
void mrbc_array_delete(mrbc_value *ary)
{
    mrbc_array *h = ary->array;
    mrbc_value *p = h->data;
    for (int i=0; i < h->n; i++, p++) {
    	mrbc_release(p);
    }
    mrbc_free(h->data);
    mrbc_free(h);
}

//================================================================
/*! resize buffer

  @param  ary	pointer to target value
  @param  size	size
  @return	mrbc_error_code
*/
__GURU__
int mrbc_array_resize(mrbc_array *h, int size)
{
	assert(size > h->size);

    mrbc_value *d2 = (mrbc_value *)mrbc_realloc(h->data, sizeof(mrbc_value) * size);
    if (!d2) return -1;

    h->data = d2;
    h->size = size;

    return 0;
}

__GURU__
int _adjust_index(mrbc_array *h, int idx, int inc)
{
    if (idx < 0) {
        idx = h->n + idx + inc;
        assert(idx>=0);
    }
    int ndx = idx;
    if ((ndx + inc) >= h->size) {	// need resize?
        ndx += inc;
    }
    if ((h->n + inc) > h->size) {
        ndx = h->n + inc;
    }
    if (ndx>idx && mrbc_array_resize(h, ndx) != 0) return -1;

    return ndx;
}
//================================================================
/*! setter

  @param  ary		pointer to target value
  @param  idx		index
  @param  set_val	set value
  @return		mrbc_error_code
*/
__GURU__
int mrbc_array_set(mrbc_value *ary, int idx, mrbc_value *set_val)
{
    mrbc_array *h = ary->array;

    idx = _adjust_index(h, idx, 0);				// adjust index if needed
    if (idx<0) return -1;						// allocation error

    if (idx < h->n) {
        mrbc_release(&h->data[idx]);			// release existing data
    }
    else {
        for(int i=h->n; i<idx; i++) {	// lazy fill here, instead of when resized
            h->data[i] = mrbc_nil_value();		// prep newly allocated cells
        }
        h->n = idx;
    }
    h->data[idx] = *set_val;					// keep the same reference count

    return 0;
}

//================================================================
/*! getter

  @param  ary		pointer to target value
  @param  idx		index
  @return		mrbc_value data at index position or Nil.
*/
__GURU__
mrbc_value mrbc_array_get(mrbc_value *ary, int idx)
{
    mrbc_array *h = ary->array;

    if (idx < 0) idx = h->n + idx;
    if (idx < 0 || idx >= h->n) return mrbc_nil_value();

    mrbc_value ret = h->data[idx];
    mrbc_retain(&ret);						// 20181029: CC Added

    return ret;
}

//================================================================
/*! push a data to tail

  @param  ary		pointer to target value
  @param  set_val	set value
  @return		mrbc_error_code
*/
__GURU__
int mrbc_array_push(mrbc_value *ary, mrbc_value *set_val)
{
    mrbc_array *h = ary->array;

    if (h->n >= h->size) {
        int size = h->size + 6;
        if (mrbc_array_resize(h, size) != 0) {
            return -1;
        }
    }
    h->data[h->n++] = *set_val;

    return 0;
}

//================================================================
/*! pop a data from tail.

  @param  ary		pointer to target value
  @return		tail data or Nil
*/
__GURU__
mrbc_value mrbc_array_pop(mrbc_value *ary)
{
    mrbc_array *h = ary->array;

    if (h->n <= 0) return mrbc_nil_value();

    return h->data[--h->n];
}

//================================================================
/*! insert a data to the first.

  @param  ary		pointer to target value
  @param  set_val	set value
  @return		mrbc_error_code
*/
__GURU__
int mrbc_array_unshift(mrbc_value *ary, mrbc_value *set_val)
{
    return mrbc_array_insert(ary, 0, set_val);
}

//================================================================
/*! removes the first data and returns it.

  @param  ary		pointer to target value
  @return		first data or Nil
*/
__GURU__
mrbc_value mrbc_array_shift(mrbc_value *ary)
{
    mrbc_array *h = ary->array;

    if (h->n <= 0) return mrbc_nil_value();

    mrbc_value ret = h->data[0];
    MEMCPY((uint8_t *)h->data, (uint8_t *)(h->data+1), sizeof(mrbc_value)*--h->n);

    return ret;
}

//================================================================
/*! insert a data

  @param  ary		pointer to target value
  @param  idx		index
  @param  set_val	set value
  @return		mrbc_error_code
*/
__GURU__
int mrbc_array_insert(mrbc_value *ary, int idx, mrbc_value *set_val)
{
    mrbc_array *h = ary->array;

    int size = _adjust_index(h, idx, 1);
    if (size < 0) return -1;

    if (idx < h->n) {			// move data
    	int blksz = sizeof(mrbc_value)*(h->n - idx);
        MEMCPY((uint8_t *)(h->data + idx + 1),(uint8_t *)(h->data + idx), blksz);	// shift
    }

    h->data[idx] = *set_val;	// set data
    h->n++;

    if (size >= h->n) {			// clear empty cells if needed
        for (int i = h->n-1; i < size; i++) {
            h->data[i] = mrbc_nil_value();
        }
        h->n = size;
    }
    return 0;
}

//================================================================
/*! remove a data

  @param  ary		pointer to target value
  @param  idx		index
  @return			mrbc_value data at index position or Nil.
*/
__GURU__
mrbc_value mrbc_array_remove(mrbc_value *ary, int idx)
{
    mrbc_array *h = ary->array;

    if (idx < 0) idx = h->n + idx;
    if (idx < 0 || idx >= h->n) return mrbc_nil_value();

    mrbc_value *p = h->data + idx;
    if (idx < --h->n) {										// shrink by 1
    	int blksz = sizeof(mrbc_value) * (h->n - idx);
        MEMCPY((uint8_t *)p, (uint8_t *)(p+1), blksz);		// shift forward
    }
    return *p;
}

//================================================================
/*! clear all

  @param  ary		pointer to target value
*/
__GURU__
void mrbc_array_clear(mrbc_value *ary)
{
    mrbc_array *h = ary->array;
    mrbc_value *p = h->data;
    for (int i=0; i < h->n; i++, p++) {
    	mrbc_release(p);                      // CC: was dec_refc 20181101
    }
    h->n = 0;
}

//================================================================
/*! compare

  @param  v1	Pointer to mrbc_value
  @param  v2	Pointer to another mrbc_value
  @retval 0	v1==v2
  @retval plus	v1 >  v2
  @retval minus	v1 <  v2
*/
__GURU__
int mrbc_array_compare(const mrbc_value *v0, const mrbc_value *v1)
{
	mrbc_value *d0 = v0->array->data;
	mrbc_value *d1 = v1->array->data;
    for (int i=0; ; i++) {
        if (i >= mrbc_array_size(v0) || i >= mrbc_array_size(v1)) {
            return mrbc_array_size(v0) - mrbc_array_size(v1);
        }
        int res = mrbc_compare(d0++, d1++);
        if (res != 0) return res;
    }
    return 0;
}

//================================================================
/*! get min, max value

  @param  ary		pointer to target value
  @param  pp_min_value	returns minimum mrbc_value
  @param  pp_max_value	returns maxmum mrbc_value
*/
__GURU__
void mrbc_array_minmax(mrbc_value *ary, mrbc_value **pp_min_value, mrbc_value **pp_max_value)
{
    mrbc_array *h = ary->array;

    if (h->n==0) {
        *pp_min_value = NULL;
        *pp_max_value = NULL;
        return;
    }
    mrbc_value *p_min_value = h->data;
    mrbc_value *p_max_value = h->data;
    mrbc_value *p           = h->data;
    for (int i = 1; i < h->n; i++, p++) {
        if (mrbc_compare(p, p_min_value) < 0) p_min_value = p;
        if (mrbc_compare(p, p_max_value) > 0) p_max_value = p;
    }
    *pp_min_value = p_min_value;
    *pp_max_value = p_max_value;
}

//================================================================
/*! method new
 */
__GURU__
void c_array_new(mrbc_value v[], int argc)
{
	mrbc_value ret;
    if (argc==0) {													// in case of new()
        ret = mrbc_array_new(0);
        if (ret.array==NULL) return;		// ENOMEM
    }
    else if (argc==1 && v[1].tt==MRBC_TT_FIXNUM && v[1].i >= 0) {	// new(num)
        ret = mrbc_array_new(v[1].i);
        if (ret.array==NULL) return;		// ENOMEM

        mrbc_value nil = mrbc_nil_value();
        if (v[1].i > 0) {
            mrbc_array_set(&ret, v[1].i - 1, &nil);
        }
    }
    else if (argc==2 && v[1].tt==MRBC_TT_FIXNUM && v[1].i >= 0) {	// new(num, value)
        ret = mrbc_array_new(v[1].i);
        if (ret.array==NULL) return;		// ENOMEM

        for (int i=0; i < v[1].i; i++) {
            mrbc_array_set(&ret, i, &v[2]);
            mrbc_retain(&v[2]);
        }
    }
    else {
    	ret = mrbc_nil_value();
        console_str("ArgumentError\n");	// raise?
    }
    SET_RETURN(ret);
}

//================================================================
/*! (operator) +
 */
__GURU__
void c_array_add(mrbc_value v[], int argc)
{
    if (GET_TT_ARG(1) != MRBC_TT_ARRAY) {
        console_str("TypeError\n");		// raise?
        return;
    }
    mrbc_array *h0 = v[0].array;
    mrbc_array *h1 = v[1].array;

    int h0sz = sizeof(mrbc_value) * h0->n;
    int h1sz = sizeof(mrbc_value) * h1->n;

    mrbc_value ret = mrbc_array_new(h0sz + h1sz);
    if (ret.array==NULL) return;		// ENOMEM

    MEMCPY((uint8_t *)(ret.array->data),        (const uint8_t *)h0->data, h0sz);
    MEMCPY((uint8_t *)(ret.array->data) + h0sz, (const uint8_t *)h1->data, h1sz);

    mrbc_value *p = ret.array->data;
    int         n = ret.array->n = h0->n + h1->n;
    for (int i=0; i<n; i++, p++) {
    	mrbc_retain(p);
    }
    mrbc_release(v+1);					// dec_refc v[1], free if not needed

    SET_RETURN(ret);
}

//================================================================
/*! (operator) []
 */
__GURU__
void c_array_get(mrbc_value v[], int argc)
{
	mrbc_value ret;
    if (argc==1 && v[1].tt==MRBC_TT_FIXNUM) {			// self[n] -> object | nil
        ret = mrbc_array_get(v, v[1].i);
    }
    else if (argc==2 &&			 						// self[idx, len] -> Array | nil
    		v[1].tt==MRBC_TT_FIXNUM &&
    		v[2].tt==MRBC_TT_FIXNUM) {
        int len = mrbc_array_size(&v[0]);
        int idx = v[1].i;
        if (idx < 0) idx += len;
        if (idx < 0) goto DONE;

        int size = (v[2].i < (len - idx)) ? v[2].i : (len - idx);
        // min(v[2].i, (len - idx))
        if (size < 0) goto DONE;

        ret = mrbc_array_new(size);
        if (ret.array==NULL) return;		// ENOMEM

        for (int i = 0; i < size; i++) {
            mrbc_value val = mrbc_array_get(v, v[1].i + i);
            mrbc_array_push(&ret, &val);
            // mrbc_retain(&val);         // CC: remove 20181029
        }
    }
    else {
        console_na("case of Array#[]");
    	ret = mrbc_nil_value();
    }
DONE:
    SET_RETURN(ret);
}

//================================================================
/*! (operator) []=
 */
__GURU__
void c_array_set(mrbc_value v[], int argc)
{
    if (argc==2 && v[1].tt==MRBC_TT_FIXNUM) {	// self[n] = val
        mrbc_array_set(v, v[1].i, &v[2]);		// raise? IndexError or ENOMEM
        v[2].tt = MRBC_TT_EMPTY;
    }
    else if (argc==3 &&							// self[n, len] = valu
    		v[1].tt==MRBC_TT_FIXNUM &&
    		v[2].tt==MRBC_TT_FIXNUM) {
        // TODO: not implement yet.
    }
    else {
        console_na("case of Array#[]=");
    }
}

//================================================================
/*! (method) clear
 */
__GURU__
void c_array_clear(mrbc_value v[], int argc)
{
    mrbc_array_clear(v);
}

//================================================================
/*! (method) delete_at
 */
__GURU__
void c_array_delete_at(mrbc_value v[], int argc)
{
    mrbc_value ret = mrbc_array_remove(v, GET_INT_ARG(1));
    SET_RETURN(ret);
}

//================================================================
/*! (method) empty?
 */
__GURU__
void c_array_empty(mrbc_value v[], int argc)
{
    int n = mrbc_array_size(v);

    SET_BOOL_RETURN(!n);
}

//================================================================
/*! (method) size,length,count
 */
__GURU__
void c_array_size(mrbc_value v[], int argc)
{
    int n = mrbc_array_size(v);

    SET_INT_RETURN(n);
}

//================================================================
/*! (method) index
 */
__GURU__
void c_array_index(mrbc_value v[], int argc)
{
    mrbc_value *value = &GET_ARG(1);
    
    mrbc_array *h = v->array;
    mrbc_value *p = h->data;
    for (int i = 0; i < h->n; i++, p++) {
        if (mrbc_compare(p, value)==0) {
            SET_INT_RETURN(i);
            return;
        }
    }
    SET_NIL_RETURN();
}

//================================================================
/*! (method) first
 */
__GURU__ void c_array_first(mrbc_value v[], int argc)
{
	mrbc_value ret = mrbc_array_get(v, 0);
	SET_RETURN(ret);
    //mrbc_retain(&val)        	// CC: removed 20181029
}

//================================================================
/*! (method) last
 */
__GURU__
void c_array_last(mrbc_value v[], int argc)
{
	mrbc_value ret = mrbc_array_get(v, -1);
	SET_RETURN(ret);
    //mrbc_retain(&val)        	// CC: removed 20181029
}

//================================================================
/*! (method) push
 */
__GURU__
void c_array_push(mrbc_value v[], int argc)
{
    mrbc_array_push(v, &v[1]);	// raise? ENOMEM
    v[1].tt = MRBC_TT_EMPTY;
}

//================================================================
/*! (method) pop
 */
__GURU__
void c_array_pop(mrbc_value v[], int argc)
{
	mrbc_value ret;
    if (argc==0) {									// pop() -> object | nil
        ret = mrbc_array_pop(v);
        SET_RETURN(ret);
    }
    else if (argc==1 && v[1].tt==MRBC_TT_FIXNUM) {	// pop(n) -> Array | nil
        // TODO: not implement yet.
    }
    else {
    	console_str("case of Array#pop");
    }
}

//================================================================
/*! (method) unshift
 */
__GURU__
void c_array_unshift(mrbc_value v[], int argc)
{
    mrbc_array_unshift(&v[0], &v[1]);	// raise? IndexError or ENOMEM
    v[1].tt = MRBC_TT_EMPTY;
}

//================================================================
/*! (method) shift
 */
__GURU__
void c_array_shift(mrbc_value v[], int argc)
{
	mrbc_value ret;
    if (argc==0) {									// shift() -> object | nil
        ret = mrbc_array_shift(v);
        SET_RETURN(ret);
    }
    else if (argc==1 && v[1].tt==MRBC_TT_FIXNUM) {	// shift() -> Array | nil
        // TODO: not implement yet.
    }
    else {
    	console_na("case of Array#shift");
    }
}

//================================================================
/*! (method) dup
 */
__GURU__
void c_array_dup(mrbc_value v[], int argc)
{
    mrbc_array *h0 = v[0].array;
    mrbc_value ret = mrbc_array_new(h0->n);
    mrbc_array *h1 = ret.array;
    if (!h1) return;		// ENOMEM

    int n = h1->n = h0->n;
    MEMCPY((uint8_t *)h1->data, (const uint8_t *)h0->data, n*sizeof(mrbc_value));

    mrbc_value *p = h1->data;
    for (int i=0; i<n; i++, p++) {
        mrbc_retain(p);
    }
    SET_RETURN(ret);
}

//================================================================
/*! (method) min
 */
__GURU__
void c_array_min(mrbc_value v[], int argc)
{
    // Subset of Array#min, not support min(n).

    mrbc_value *p_min_value, *p_max_value;

    mrbc_array_minmax(&v[0], &p_min_value, &p_max_value);
    if (p_min_value==NULL) {
        SET_NIL_RETURN();
    }
    else {
    	SET_RETURN(*p_min_value);
    	mrbc_retain(p_min_value);       // CC: 20181029 needed?
    }
}

//================================================================
/*! (method) max
 */
__GURU__
void c_array_max(mrbc_value v[], int argc)
{
    // Subset of Array#max, not support max(n).

    mrbc_value *p_min_value, *p_max_value;

    mrbc_array_minmax(&v[0], &p_min_value, &p_max_value);
    if (p_max_value==NULL) {
        SET_NIL_RETURN();
    }
    else {
    	SET_RETURN(*p_max_value);
    	mrbc_retain(p_max_value);        // CC: 20181029 needed?
    }
}

//================================================================
/*! (method) minmax
 */
__GURU__
void c_array_minmax(mrbc_value v[], int argc)
{
    // Subset of Array#minmax, not support minmax(n).

    mrbc_value *p_min_value, *p_max_value;
    mrbc_value nil = mrbc_nil_value();
    mrbc_value ret = mrbc_array_new(2);

    mrbc_array_minmax(&v[0], &p_min_value, &p_max_value);
    if (p_min_value==NULL) p_min_value = &nil;
    if (p_max_value==NULL) p_max_value = &nil;

    mrbc_array_set(&ret, 0, p_min_value);
    mrbc_array_set(&ret, 1, p_max_value);
    mrbc_retain(p_min_value);				// needed?
    mrbc_retain(p_max_value);				// needed?

    SET_RETURN(ret);
}

#if MRBC_USE_STRING
__GURU__
void _rfc(mrbc_value *str, mrbc_value *v)
{
	char buf[8];
	guru_sprintf(buf, "^%d_", v->self->refc);
	mrbc_string_append_cstr(str, buf);
}
//================================================================
/*! (method) inspect
 */
__GURU__
void c_array_inspect(mrbc_value v[], int argc)
{
	mrbc_value *ary = v+argc;
	mrbc_value ret  = mrbc_string_new("[");
    if (!ret.str) {
    	SET_NIL_RETURN();
    	return;
    }
    for (int i = 0; i < mrbc_array_size(v); i++) {
        if (i != 0) mrbc_string_append_cstr(&ret, ", ");
        mrbc_value vi = mrbc_array_get(v, i);
        mrbc_value s  = mrbc_send(ary, &vi, "inspect", 0);
        mrbc_string_append(&ret, &s);
        mrbc_release(&s);           		// CC: added 20181029
        mrbc_release(&vi);                 	// CC: added 20181029
    }
    mrbc_string_append_cstr(&ret, "]");

    SET_RETURN(ret);
}

//================================================================
/*! (method) join
 */
__GURU__
void c_array_join_1(mrbc_value v[], int argc,
                    mrbc_value *src, mrbc_value *ret, mrbc_value *separator)
{
    if (mrbc_array_size(src)==0) return;

    int i = 0;
    int error = 0;
    while (!error) {
        if (src->array->data[i].tt==MRBC_TT_ARRAY) {
            c_array_join_1(v, argc, &src->array->data[i], ret, separator);
        }
        else {
            mrbc_value s1 = mrbc_send(v+argc, &src->array->data[i], "inspect", 0);
            error |= mrbc_string_append(ret, &s1);
            mrbc_string_delete(&s1);
        }
        if (++i >= mrbc_array_size(src)) break;	// normal return.
        error |= mrbc_string_append(ret, separator);
    }
}

__GURU__
void c_array_join(mrbc_value v[], int argc)
{
    mrbc_value ret = mrbc_string_new(NULL);
    if (!ret.str) {
        SET_NIL_RETURN();
        return;
    }
    mrbc_value separator = (argc==0)
    		? mrbc_string_new("")
    		: mrbc_send(v+argc, &v[1], "inspect", 0);

    c_array_join_1(v, argc, &v[0], &ret, &separator);
    mrbc_dec_refc(&separator);

    SET_RETURN(ret);
}
#endif

//================================================================
/*! initialize
 */
__GURU__
void mrbc_init_class_array()
{
    mrbc_class *c = mrbc_class_array = mrbc_define_class("Array", mrbc_class_object);

    mrbc_define_method(c, "new",       c_array_new);
    mrbc_define_method(c, "+",         c_array_add);
    mrbc_define_method(c, "[]",        c_array_get);
    mrbc_define_method(c, "at",        c_array_get);
    mrbc_define_method(c, "[]=",       c_array_set);
    mrbc_define_method(c, "<<",        c_array_push);
    mrbc_define_method(c, "clear",     c_array_clear);
    mrbc_define_method(c, "delete_at", c_array_delete_at);
    mrbc_define_method(c, "empty?",    c_array_empty);
    mrbc_define_method(c, "size",      c_array_size);
    mrbc_define_method(c, "length",    c_array_size);
    mrbc_define_method(c, "count",     c_array_size);
    mrbc_define_method(c, "index",     c_array_index);
    mrbc_define_method(c, "first",     c_array_first);
    mrbc_define_method(c, "last",      c_array_last);
    mrbc_define_method(c, "push",      c_array_push);
    mrbc_define_method(c, "pop",       c_array_pop);
    mrbc_define_method(c, "shift",     c_array_shift);
    mrbc_define_method(c, "unshift",   c_array_unshift);
    mrbc_define_method(c, "dup",       c_array_dup);
    mrbc_define_method(c, "min",       c_array_min);
    mrbc_define_method(c, "max",       c_array_max);
    mrbc_define_method(c, "minmax",    c_array_minmax);
#if MRBC_USE_STRING
    mrbc_define_method(c, "inspect",   c_array_inspect);
    mrbc_define_method(c, "to_s",      c_array_inspect);
    mrbc_define_method(c, "join",      c_array_join);
#endif
}
