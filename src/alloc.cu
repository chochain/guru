#include "hip/hip_runtime.h"
/*! @file
  @brief
  mrubyc memory management.

  <pre>
  Copyright (C) 2015-2018 Kyushu Institute of Technology.
  Copyright (C) 2015-2018 Shimane IT Open-Innovation Center.

  This file is distributed under BSD 3-Clause License.

  Memory management for objects in mruby/c.

  </pre>
*/
#include <stdio.h>
#include <assert.h>
#include "alloc.h"

// TLSF: Two-Level Segregated Fit allocator with O(1) time complexity.
// Layer 1st(f), 2nd(s) model, and ignored last 4bit (i.e. smallest block is 16-bytes)
// f : size
// 0 : 0000-007f
// 1 : 0080-00ff
// 2 : 0100-01ff
// 3 : 0200-03ff
// 4 : 0400-07ff
// 5 : 0800-0fff
// 6 : 1000-1fff
// 7 : 2000-3fff
// 8 : 4000-7fff
// 9 : 8000-ffff

#ifndef L1_BITS			// 0000 0000 0000 0000
#define L1_BITS 	9	// ~~~~~~~~~~~
#define L1_MASK 	((1<<L1_BITS)-1)
#endif
#ifndef L2_BITS			// 0000 0000 0000 0000
#define L2_BITS 	3	//            ~~~
#define L2_MASK 	((1<<L2_BITS)-1)
#endif
#ifndef XX_BITS			// 0000 0000 0000 0000
#define XX_BITS 	4	//                ~~~~
#define XX_BLOCK	(1 << XX_BITS)					// 16-bytes
#endif

#define L1(i) 			(((i) >> L2_BITS) & L1_MASK)
#define L2(i) 			((i) & L2_MASK)
#define MSB_BIT 		0x8000
#define L1_MAP(i)      	(MSB_BIT >> L1(i))
#define L2_MAP(i)		(MSB_BIT >> L2(i))

// free memory block index
#define BLOCK_SLOTS		((L1_BITS + 1) * (1 << L2_BITS))

#define NEXT(p) 		((uint8_t *)(p) + (p)->size)
#define PREV(p) 		((uint8_t *)(p) - (p)->offset)
#define OFF(p0,p1) 		((uint8_t *)(p1) - (uint8_t *)(p0))

// memory pool
__GURU__ unsigned int 	_memory_pool_size;
__GURU__ uint8_t     	*_memory_pool;

__GURU__ free_block 	*_free_list[BLOCK_SLOTS + 1];

// free memory bitmap
__GURU__ uint16_t 		_l1_map;
__GURU__ uint16_t 		_l2_map[L1_BITS + 2]; 		// + sentinel

#define GET_L1(i)		(_l1_map & (L1_MAP(i) - 1))
#define SET_L1(i)		(_l1_map |= L1_MAP(i))
#define CLR_L1(i)	    (_l1_map &= ~L1_MAP(i))
#define L2_KEY(i)		(_l2_map[L1(i)])
#define GET_L2(i)		(L2_KEY(i) & (L2_MAP(i) - 1))
#define SET_L2(i)	    (L2_KEY(i) |= L2_MAP(i))
#define CLR_L2(i)		(L2_KEY(i) &= ~L2_MAP(i))
#define CLEAR_MAP(i)	{ CLR_L2(i); if (L2_KEY(i)==0) CLR_L1(i); }
//================================================================
/*! Number of leading zeros.

  @param  x	target (16bit unsined)
  @retval int	nlz value
*/
__GURU__ int
__nlz16(uint16_t x)
{
    if (x==0) return 16;

    int n = 1;
    if ((x>> 8)==0) { n+=8; x<<=8; }
    if ((x>>12)==0) { n+=4; x<<=4; }
    if ((x>>14)==0) { n+=2; x<<=2; }

    return n - (x>>15);
}

__GURU__ __INLINE__ int
__calc_index(int l1, int l2)
{
    assert(l1 >= 0);
    assert(l1 <= L1_BITS);
    assert(l2 >= 0);
    assert(l2 <= L2_MASK);

    return (l1 << L2_BITS) + l2;
}

//================================================================
/*! calc f and s, and returns fli,sli of free_blocks

  @param  alloc_size	alloc size
  @retval int		index of free_blocks
*/
__GURU__ int
_get_index(unsigned int alloc_size)
{
    if ((alloc_size >> (L1_BITS+L2_BITS+XX_BITS)) != 0) {		// overflow check
        return BLOCK_SLOTS;
    }
    int l1    = 16 - __nlz16(alloc_size >> (L2_BITS + XX_BITS));	// 1st level index
    int shift =	l1 + XX_BITS - ((l1==0) ? 0 : 1);
    int l2    = (alloc_size >> shift) & L2_MASK;					// 2nd level index

    return __calc_index(l1, l2);
}

//================================================================
/*! just remove the free_block *target from index

  @param  target	pointer to target block.
*/
__GURU__ void
_remove_index(free_block *target)
{
    if (target->prev==NULL) {	// head of linked list?
        int index = _get_index(target->size) - 1;

        if ((_free_list[index]=target->next)==NULL) {
            CLEAR_MAP(index);
        }
    }
    else {	// link previous to next
        target->prev->next = target->next;
    }
    if (target->next != NULL) {	// reverse link
        target->next->prev = target->prev;
    }
}

//================================================================
/*! Mark that block free and register it in the free index table.

  @param  target	Pointer to target block.
*/
__GURU__ void
_mark_free(free_block *target)
{
    target->free = FLAG_FREE_BLOCK;

    int index = _get_index(target->size) - 1;

#ifdef MRBC_DEBUG
    int l1 = L1(index);  					// debug: (index>>3) & 0xff
    int l2 = L2(index);  					// debug: index & 0x7
    free_block *blk = _free_list[index];	// debug:
#endif

    SET_L1(index);							// update maps
    SET_L2(index);

    target->next = _free_list[index];		// current block
    target->prev = NULL;
    if (target->next != NULL) {				// non-end block
        target->next->prev = target;
    }
    _free_list[index] = target;				// keep target as last block
}

//================================================================
/*! merge ptr1 and ptr2 block.
  ptr2 will disappear

  @param  ptr1	pointer to free block 1
  @param  ptr2	pointer to free block 2
*/
__GURU__ void
_merge_blocks(free_block *p0, free_block *p1)
{
    assert(p0 < p1);

    // merge ptr1 and ptr2
    p0->tail  = p1->tail;
    p0->size += p1->size;

    // update block info
    if (p0->tail==FLAG_NOT_TAIL_BLOCK) {
        free_block *next = (free_block *)NEXT(p0);
        next->offset = OFF(p0, next);
    }
}

__GURU__ void
_merge_with_next(free_block *target)
{
	if (target->tail!=FLAG_NOT_TAIL_BLOCK) return;

	free_block *next = (free_block *)NEXT(target);

	if (next->free!=FLAG_FREE_BLOCK) return;

	_remove_index(next);
	_merge_blocks(target, next);
}

__GURU__ free_block*
_merge_with_prev(free_block *target)
{
    free_block *prev = (free_block *)PREV(target);

    if (prev==NULL || prev->free!=FLAG_FREE_BLOCK) return target; 	// no change

    _remove_index(prev);
    _merge_blocks(prev, target);

    return prev;
}

//================================================================
/*! Split block by size

  @param  target	pointer to target block
  @param  size	size
  @retval NULL	no split.
  @retval FREE_BLOCK *	pointer to splitted free block.
*/
__GURU__ free_block*
_split_free_block(free_block *target, unsigned int size, int merge)
{
    if (target->size < (size + sizeof(free_block) + XX_BLOCK)) {
    	return NULL;	// out of memory!
    }

    // split block, free
    free_block *free = (free_block *)((uint8_t *)target + size);	// future next block
    free_block *next = (free_block *)NEXT(target);					// current next

    free->size   = target->size - size;
    free->offset = OFF(target, free);
    free->tail   = target->tail;

    target->size = size;
    target->tail = FLAG_NOT_TAIL_BLOCK;

    if (free->tail==FLAG_NOT_TAIL_BLOCK) {
        next->offset = OFF(free, next);
    }
    if (free != NULL) {
    	if (merge) _merge_with_next(free);
    	_mark_free(free);
    }
    return free;
}

__GURU__ int
_get_free_index(unsigned int alloc_size)
{
    int index = _get_index(alloc_size);	// find free memory block

    if (_free_list[index] != NULL) {
    	return index;					// allocated before, keep using the same block
    }

    // no previous block exist, create a new one
    int l1 = L1(index);
    int l2 = L2(index);

    int used = GET_L2(index);
    if (used) {							// check any 2nd level available
    	l2 = __nlz16(used);
    }
    else {								// go up to 1st level
    	used = GET_L1(index);
        if (used) {						// allocate new 1st & 2nd level indices
        	l1 = __nlz16(used);			// CC: this might have problem, 20181104 because used is changed
            l2 = __nlz16(_l2_map[l1]);
        }
        else return -1;					// out of memeory
    }
    return __calc_index(l1, l2);		// new index
}

/*
 * TODO: refactor into _remove_index()
 */
__GURU__ free_block*
_mark_used(int index)
{
    free_block *target = _free_list[index];

    assert(target!=NULL);

    // remove free_blocks index
    target->free      = FLAG_USED_BLOCK;
    _free_list[index] = target->next;

    if (target->next==NULL) {					// top of linked list
        CLEAR_MAP(index);						// release the index
    }
    else {
    	target->next->prev = target->prev;		// CC: is this needed? 20181104
    }
    return target;
}

//================================================================
/*! initialize

  @param  ptr	pointer to free memory block.
  @param  size	size. (max 64KB. see mrbc_memsize_t)
*/
__GURU__ void
_init_mmu(void *mem, unsigned int size)
{
    assert(size != 0);
    assert(size <= (mrbc_memsize_t)(~0));

    _memory_pool      = (uint8_t *)mem;
    _memory_pool_size = size;

    // initialize entire memory pool as the first block
    free_block *block  = (free_block *)_memory_pool;
    block->tail   = FLAG_TAIL_BLOCK;
    block->free   = FLAG_FREE_BLOCK;
    block->size   = _memory_pool_size;
    block->offset = 0;

    _mark_free(block);
}

//================================================================
/*! allocate memory

  @param  size	request size.
  @return void * pointer to allocated memory.
  @retval NULL	error.
*/
__GURU__ void*
mrbc_alloc(unsigned int size)
{
    // TODO: maximum alloc size
    //  (1 << (L1_BITS + L2_BITS + XX_BITS)) - alpha
    unsigned int alloc_size = size + sizeof(free_block);

    alloc_size += ((8 - alloc_size) & 7);	// 8-byte align

    // check minimum alloc size. if need.
#if 0
    if (alloc_size < XX_BLOCK) {
        alloc_size = XX_BLOCK;
    }
#else
    assert(alloc_size >= XX_BLOCK);
#endif

	int index = _get_free_index(alloc_size);
    free_block *target = _mark_used(index);

    assert(target->size >= alloc_size);
    assert(((uintptr_t)target & 7)==0);							// check alignment

    // split the allocated block
    if (!_split_free_block(target, alloc_size, 0)) return NULL;	// out of memory?

#ifdef MRBC_DEBUG
    uint8_t *p = (uint8_t *)target + sizeof(used_block);
    for (int i=0; i<target->size-sizeof(used_block); i++) *p++ = 0xaa;
#endif
    return (uint8_t *)target + sizeof(used_block);
}

//================================================================
/*! re-allocate memory

  @param  ptr	Return value of mrbc_raw_alloc()
  @param  size	request size
  @return void * pointer to allocated memory.
  @retval NULL	error.
*/
__GURU__ void*
mrbc_realloc(void *ptr, unsigned int size)
{
    used_block *target      = (used_block *)((uint8_t *)ptr - sizeof(used_block));
    unsigned int alloc_size = size + sizeof(free_block);

    // align 4 byte
    alloc_size += ((8 - alloc_size) & 7);					// CC: 20181030 from 4 to 8-byte align

    // expand part1.
    // next phys block is free and check enough size?
    if (alloc_size > target->size) {
    	_merge_with_next((free_block *)target);
    }
    if (alloc_size==target->size) {								// is the size the same now?
        return ptr;
    }
    if (alloc_size < target->size) {							// need to split
        _split_free_block((free_block *)target, alloc_size, 1);
        return ptr;
    }

    // expand part2. new alloc and deep copy
    void *new_ptr = mrbc_alloc(size);
    if (!new_ptr) return NULL;  								// ENOMEM

    uint8_t *d = (uint8_t *)new_ptr, *s = (uint8_t *)ptr;
    for (int i=0; i < (target->size-sizeof(used_block)); i++, *d++=*s++);

    mrbc_free(ptr);

    return new_ptr;
}

//================================================================
/*! release memory

  @param  ptr	Return value of mrbc_raw_alloc()
*/
__GURU__ void
mrbc_free(void *ptr)
{
    // get target block
    free_block *target = (free_block *)((uint8_t *)ptr - sizeof(used_block));

    _merge_with_next(target);
    _mark_free(_merge_with_prev(target));	// target, add to index
}

//================================================================
/*! release memory, vm used.

  @param  vm	pointer to VM.
*/
__GURU__ void
mrbc_free_all()
{
    used_block *p = (used_block *)_memory_pool;
    while (1) {
    	if (p->free==FLAG_USED_BLOCK) {
    		mrbc_free((uint8_t *)p + sizeof(used_block));
    	}
    	if (p->tail==FLAG_TAIL_BLOCK) break;
    	p = (used_block *)NEXT(p);
    }
}

#ifdef MRBC_DEBUG
//================================================================
/*! statistics

  @param  *total	returns total memory.
  @param  *used		returns used memory.
  @param  *free		returns free memory.
  @param  *fragment	returns memory fragmentation
*/
__global__ void
_guru_alloc_stat(int v[])
{
	if (threadIdx.x!=0 || blockIdx.x!=0) return;

    int total = 0;
    int nfree = 0;
    int free  = 0;
    int nused = 0;
    int used  = 0;
    int nblk  = 0;
    int nfrag = 0;

    used_block *p = (used_block *)_memory_pool;
    
    int flag = p->free;
    while (1) {
        if (flag != p->free) {       // supposed to be merged
            nfrag++;
            flag = p->free;
        }

        total += p->size;
        nblk  += 1;
        if (p->free==FLAG_FREE_BLOCK) {
        	nfree += 1;
        	free  += p->size;
        }
        if (p->free==FLAG_USED_BLOCK) {
        	nused += 1;
        	used  += p->size;
        }

        if (p->tail==FLAG_TAIL_BLOCK) break;

        p = (used_block *)NEXT(p);
    }
    v[0] = total;
    v[1] = nfree;
    v[2] = free;
    v[3] = nused;
    v[4] = used;
}

__global__ void guru_memory_init(void *ptr, unsigned int sz)
{
	if (threadIdx.x!=0 || blockIdx.x!=0) return;

	_init_mmu(ptr, sz);
}

__host__ void *
guru_malloc(size_t sz, int type)
{
	void *mem;

	switch (type) {
	case 0: 	hipMalloc(&mem, sz); break;			// allocate device memory
	default: 	hipMallocManaged(&mem, sz);			// managed (i.e. paged) memory
	}
    if (hipSuccess != hipGetLastError()) return NULL;

    return mem;
}

__host__ void
dump_alloc_stat(void)
{
	int *v;
	hipMallocManaged(&v, 8*sizeof(int));

	_guru_alloc_stat<<<1,1>>>(v);
	hipDeviceSynchronize();

	printf("\ttotal %d(0x%x)> free=%d(%d), used=%d(%d), %d%% allocated\n",
				v[0], v[0], v[1], v[2], v[3], v[4], (int)(100*(v[4]+1)/v[0]));

	hipFree(v);
}
#endif
