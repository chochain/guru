#include "hip/hip_runtime.h"
/*! @file
  @brief
  GURU bytecode loader (host load IREP code, build image and copy into CUDA memory).

  alternatively, load_gpu.cu can be used for device image building
  <pre>
  Copyright (C) 2019- Greeni

  This file is distributed under BSD 3-Clause License.

  </pre>
*/
#include <assert.h>
#include "vm_config.h"

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>

#include "guru.h"
#include "mmu.h"
#include "state.h"
#include "errorcode.h"

#include "load.h"

#if GURU_HOST_IMAGE
//================================================================
/*!@brief
  Parse header section.

  @param  vm    A pointer of VM.
  @param  pos	A pointer of pointer of RITE header.
  @return int	zero if no error.

  <pre>
  Structure
  "RITE"	identifier
  "0004"	version
  0000		CRC
  0000_0000	total size
  "MATZ"	compiler name
  "0000"	compiler version
  </pre>
*/
__HOST__ U32
bin_to_u32(const void *s)
{
    U32 x = *((U32*)s);
    return (x << 24) | ((x & 0xff00) << 8) | ((x >> 8) & 0xff00) | (x >> 24);
}

//================================================================
/*!@brief
  Get 16bit value from memory big endian.

  @param  s	Pointer of memory.
  @return	16bit unsigned value.
*/
__HOST__ U16
bin_to_u16(const void *s)
{
    U16 x = *((U16 *)s);
    return (x << 8) | (x >> 8);
}

__HOST__ int
_check_header(U8 **pos)
{
    const U8 *p = *pos;

    if (memcmp(p, "RITE000", 7)==0) {
    	// Rite binary version
    	// 0002: mruby 1.0
    	// 0003: mruby 1.1, 1.2
    	// 0004: mruby 1.3, 1.4
    	// 0005: mruby 2.0
    	U8 c = *(p+7);
        if (c < '3' || c > '4') {
        	return LOAD_FILE_HEADER_ERROR_VERSION;
        }
    }
    /* Ignore CRC */
    /* Ignore size */
    if (memcmp(p + 14, "MATZ", 4) != 0) {
        return LOAD_FILE_HEADER_ERROR_MATZ;
    }
    // Rite VM version
    // 0000: mruby 1.x
    // 0002: mruby 2.x
    if (memcmp(p + 18, "0000", 4) != 0) {
        return LOAD_FILE_HEADER_ERROR_VERSION;
    }
    *pos += 22;

    return NO_ERROR;
}

//
// building memory image, offset-based with alignment
//
__HOST__ void
_to_gv(GV v[], U32 n, U8 *p, bool sym)
{
    // build POOL or SYM block
    char buf[64+1];
    for (U32 i=0; i < n; i++, v++) {
        U32  tt = sym ? 3 : *p++;
        U32  len = bin_to_u16(p);	p += sizeof(U16);

        switch (tt) {
        case 0:	// String
        	v->raw = p;
        	v->gt  = GT_STR;
        	break;
        case 1: // Integer (31-bit)
            memcpy(buf, p, len);
            buf[len] = '\0';
            v->i   = atoi(buf);
            v->gt  = GT_INT;
            break;
        case 2: // Float (32-bit)
            memcpy(buf, p, len);
            buf[len] = '\0';
            v->f   = (float)atof(buf);		// atof() returns double
            v->gt  = GT_FLOAT;
            break;
        case 3: // Symbol
        	v->raw = p;
        	v->gt  = GT_SYM;
        	break;
        default: // Others (not yet supported)
        	v->gt   = GT_NIL;
        	v->self = NULL;
        	break;
        }
        p += len + (sym ? 1 : 0);
    }
}

//================================================================
/*!@brief
  read one irep section.

  @param  vm    A pointer of VM.
  @param  pos	A pointer of pointer of IREP section.
  @return       Pointer of allocated IREP or NULL

  <pre>
  (loop n of child irep bellow)
  0000_0000	record size
  0000		n of local variable
  0000		n of register
  0000		n of child irep

  0000_0000	n of byte code  (ISEQ BLOCK)
  ...		byte codes		(padded to 4-byte align)

  0000_0000	n of pool		(POOL BLOCK)
  (loop n of pool)
  00		type
  0000		length
  ...		pool data

  0000_0000	n of symbol		(SYMS BLOCK)
  (loop n of symbol)
  0000		length
  ...		symbol data
  </pre>
*/
__HOST__ guru_irep*
_build_image(U8 **src)							// pos will be advance to next IREP block
{
	guru_irep irep;
    U8  *p = *src;

    // Header: sz, nlocals, nregs, rlen
    irep.size 	= bin_to_u32(p); 		p += sizeof(U32);			// IREP size
    irep.nv 	= bin_to_u16(p);		p += sizeof(U16);			// number of local variables
    irep.nr 	= bin_to_u16(p);		p += sizeof(U16);			// number of registers used
    irep.r  	= bin_to_u16(p);		p += sizeof(U16);			// number of child IREP blocks

    // ISEQ block
    irep.i 		= bin_to_u32(p);		p += sizeof(U32);			// ISEQ (bytecodes) length
    U8 *iseq    = (p += -(U32A)p & 3);								// ISEQ block (32-bit aligned)
    U32 iseq_sz = sizeof(U32)*irep.i;	p += iseq_sz;				// skip ISEQ (code) block
    U32 reps_sz = sizeof(guru_irep *) * irep.r;						// child REPS block
    U32 img_sz  = sizeof(guru_irep) + iseq_sz + reps_sz;
    guru_irep *tgt = (guru_irep *)cuda_malloc(ALIGN64(img_sz), 1);	// target CUDA IREP image (managed mem)
    assert(tgt);

#if GURU_DEBUG
    memset(tgt, 0xaa, img_sz);
#endif // GURU_DEBUG

    memcpy(tgt, &irep, sizeof(guru_irep));							// dup IREP header fields
    memcpy(U8PADD(tgt, sizeof(guru_irep)), iseq,  iseq_sz);			// copy ISEQ block
    tgt->size = img_sz;
    tgt->reps = (RIrep **)U8PADD(tgt, sizeof(guru_irep)+ALIGN(iseq_sz)); // pointer to child REPS

    // POOL block
    tgt->p   = bin_to_u32(p);			p += sizeof(U32);			// pool element count
    U8 *pool = p;
    for (U32 i=0; i<tgt->p; i++) {									// 1st pass (skim through pool)
    	U32 len = bin_to_u16(++p);		p += sizeof(U16)+len;
    }
    // SYM block
    tgt->s = bin_to_u32(p);				p += sizeof(U32);			// symbol element count
    U8 *sym = p;
    for (U32 i=0; i<tgt->s; i++) {									// 1st pass (skim through sym)
    	U32 len = bin_to_u16(p)+1;		p += sizeof(U16)+len;
    }
    *src = p;														// return source pointer

    // prep Register File block which combines Reps, Pooled objects & Symbol table
    U32 pool_sz = sizeof(GV) * (tgt->p + tgt->s);
    U8 *blk = (img_sz + pool_sz < CUDA_MIN_MEMBLOCK_SIZE)			// CUDA alloc 0x200B min
    	? U8PADD(tgt, img_sz)										// utilize free space if any
    	: (U8*)cuda_malloc(pool_sz, 1);
    assert(blk);

#if GURU_DEBUG
    memset(blk, 0xaa, pool_sz);
#endif // GURU_DEBUG

    tgt->pool = (GV *)blk;
    _to_gv(tgt->pool, 			tgt->s, sym,  1);					// symbol table 1st  (faster)
    _to_gv(tgt->pool + tgt->s,  tgt->p, pool, 0);					// pooled object 2nd (one extra calc)

    return tgt;														// position pointer ends here
}
//================================================================
/*!@brief
  Parse IREP section.

  @param  vm    A pointer of VM.
  @param  pos	A pointer of pointer of IREP section.
  @return       Pointer of allocated IREP or NULL

  <pre>
  Structure
  "IREP"	section identifier
  0000_0000	section size
  "0000"	rite version
  </pre>
*/
__HOST__ guru_irep*
_load_irep(U8 **src)
{
	guru_irep *irep = _build_image(src);			// build CUDA image (in managed memory) from host image

    // recursively create the child irep tree
    for (U32 i=0; i < irep->r; i++) {				// number of irep children
    	irep->reps[i] = _load_irep(src);			// load a child irep recursively (from host image)
    }
    return irep;		// a pointer to CUDA irep (in managed memory)
}

//================================================================
/*!@brief
  Parse LVAR section.

  @param  vm    A pointer of VM.
  @param  pos	A pointer of pointer of LVAR section.
  @return int	zero if no error.
*/
__HOST__ U32
_load_lvar(U8 **pos)
{
    U8  *p     = *pos;
    U32 sec_sz = bin_to_u32(p+sizeof(U32));

    // TODO: local variable is not supported yet

    *pos += sec_sz;

    return NO_ERROR;
}

//================================================================
/*!@brief
  Load the VM bytecode.

  @param  vm    Pointer to VM.
  @param  ptr	Pointer to bytecode.

*/
__HOST__ U8 *
guru_parse_bytecode(U8 *src)
{
	U8  **sp = (U8 **)&src;			// a pointer to pointer, so that we can pass and adjust the pointer
	int ret  = _check_header(sp);

	U8 *irep;
    while (ret==NO_ERROR) {
        if (memcmp(*sp, "IREP", 4)==0) {
        	*sp += 4 + sizeof(U32);								// skip "IREP", irep_sz
            if (memcmp(*sp, "0000", 4) != 0) break;				// IREP version
            *sp += 4;											// skip "0000"

        	ret = ((irep = (U8*)_load_irep(sp))==NULL)
        			? LOAD_FILE_IREP_ERROR_ALLOCATION
        			: NO_ERROR;
        }
        else if (memcmp(*sp, "LVAR", 4)==0) {
            ret = _load_lvar(sp);
        }
        else if (memcmp(*sp, "END\0", 4)==0) {
            break;
        }
    }
    return (ret==NO_ERROR) ? irep : NULL;
}
#endif 	// GURU_HOST_IMAGE
