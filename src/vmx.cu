/*! @file
  @brief
  GURU VM pool implementation

  <pre>
  Copyright (C) 2019- GreenII

  This file is distributed under BSD 3-Clause License.

  initialize VM
  	  allocate vm cuda memory
  execute
      start main loop
         invoke every VM (in serial or parallel)
  </pre>
*/
#include <pthread.h>

#include "guru.h"
#include "util.h"
#include "mmu.h"
#include "vm.h"
#include "vmx.h"
#include "state.h"
#include "load.h"
#include "debug.h"

#define _LOCK		(pthread_mutex_lock(&_mutex))
#define _UNLOCK		(pthread_mutex_unlock(&_mutex))

__GPU__ void _vm_init(VM *vm, int i, int step)
{
	if (blockIdx.x!=0 || threadIdx.x!=0) return;

	vm->init(i, step);
}

__GPU__ void _vm_exec(VM *vm)
{
	if (blockIdx.x!=0 || threadIdx.x!=0) return;

	vm->exec();

	if (vm->run!=VM_STATUS_STOP) return;

	StateMgr *sm = new StateMgr(vm);					// needs a helper
	sm->free_states();
}

__GPU__ void _vm_prep(VM *vm, U8 *u8_gr)
{
	if (blockIdx.x!=0 || threadIdx.x!=0) return;

	vm->prep(u8_gr);
}

class VM_Pool::Impl
{
	VM 	*_vm_pool = NULL;
	U32	_vm_cnt   = 0;

	pthread_mutex_t _mutex;
	hipStream_t _st_pool[MIN_VM_COUNT];

	int
	_has_job()
	{
    	VM *vm = _vm_pool;
    	for (U32 i=0; i<MIN_VM_COUNT; i++, vm++) {
    		if (vm->run==VM_STATUS_RUN && !vm->err) return 1;
    	}
    	return 0;
	}

	__HOST__ int
	_set_status(U32 mid, U32 new_status, U32 status_flag)
	{
		VM *vm = &_vm_pool[mid];
		if (!(vm->run & status_flag)) return -1;		// transition state machine

		_LOCK;
		vm->run = new_status;
		_UNLOCK;

		return 0;
	}

	__HOST__ int ready(U32 mid) { return _set_status(mid, VM_STATUS_RUN,  VM_STATUS_READY); }
	__HOST__ int hold(U32 mid)  { return _set_status(mid, VM_STATUS_HOLD, VM_STATUS_RUN);   }
	__HOST__ int stop(U32 mid)  { return _set_status(mid, VM_STATUS_STOP, VM_STATUS_RUN);   }

public:
	Impl(U32 step)
	{
		VM *vm = _vm_pool = (VM*)cuda_malloc(sizeof(VM) * MIN_VM_COUNT, 1);

		for (U32 i=0; i<MIN_VM_COUNT; i++, vm++) {
			hipStreamCreateWithFlags(&_st_pool[i], hipStreamNonBlocking);
			_vm_init<<<1,1, 0, _st_pool[i]>>>(vm, i, step);
		}
		GPU_SYNC();
	}

	~Impl()
	{
		for (U32 i=0; i<MIN_VM_COUNT; i++) {
			hipStreamDestroy(_st_pool[i]);
		}
	}

	__HOST__ S32
	vm_main_start()
	{
		// TODO: spin off as a server thread
		do {
			VM *vm = (VM*)_vm_pool;
			for (U32 i=0; i<MIN_VM_COUNT; i++, vm++) {		// TODO: parallel
				if (!vm->state || vm->run!=VM_STATUS_RUN) continue;
				// add pre-hook here
				if (debug_disasm((guru_vm*)vm)) {
					vm->err = 1;							// stop a run-away loop
				}
				else {
					_vm_exec<<<1,1,0,_st_pool[i]>>>(vm);	// guru -x to run without single-stepping
				}
				hipError_t e = hipGetLastError();
				if (e) {
					printf("CUDA ERROR: %s, bailing\n", hipGetErrorString(e));
					vm->err = 1;
				}
				// add post-hook here
			}
			GPU_SYNC();										// TODO: cooperative thread group
#if GURU_USE_CONSOLE
			guru_console_flush(ses->out, ses->trace);		// dump output buffer
#endif  // GURU_USE_CONSOLE
		} while (_has_job());								// join()

		return 0;
	}

	__HOST__ S32
	vm_get(U8 *ibuf)
	{
		if (!_vm_pool) 				return -1;
		if (_vm_cnt>=MIN_VM_COUNT) 	return -1;

		VM *vm = &_vm_pool[_vm_cnt];

#if GURU_HOST_GRIT_IMAGE
		U8 *gr = (U8*)parse_bytecode(ibuf);
		if (!gr) return -2;

		_vm_prep<<<1,1>>>(vm, gr);
#else
		_prep<<<1,1,0,vm->st>>>(vm, ibuf);				// acquire VM, vm status will changed
#endif // GURU_HOST_GRIT_IMAGE
		GPU_SYNC();

		debug_vm_irep((guru_vm*)&vm);
		ready(_vm_cnt);

		return _vm_cnt++;
	}
};

__HOST__ VM_Pool::VM_Pool(int step) : _impl(new Impl((U32)step)) {}
__HOST__ VM_Pool::~VM_Pool() = default;

__HOST__ int
VM_Pool::start()
{
	return _impl->vm_main_start();
}

__HOST__ int
VM_Pool::get(char *ibuf)
{
	return _impl->vm_get((U8*)ibuf);
}
