#include "hip/hip_runtime.h"
/*! @file
 *
  @brief
  GURU Integer and Float class

  <pre>
  Copyright (C) 2019- GreenII.

  This file is distributed under BSD 3-Clause License.

  </pre>
*/
#include "guru.h"
#include "value.h"
#include "static.h"
#include "c_fixnum.h"

#if GURU_USE_STRING
#include "c_string.h"
#include "puts.h"
#endif

//================================================================
/*! (operator) [] bit reference
 */
__GURU__ void
c_int_bitref(GV v[], U32 argc)
{
    if (0 <= v[1].i && v[1].i < 32) {
        RETURN_INT((v[0].i & (1 << v[1].i)) ? 1 : 0);
    }
    else {
        RETURN_INT(0);
    }
}

//================================================================
/*! (operator) unary -
 */
__GURU__ void
c_int_negative(GV v[], U32 argc)
{
    GI n = ARG_INT(0);
    RETURN_INT(-n);
}

//================================================================
/*! (operator) ** power
 */
__GURU__ void
c_int_power(GV v[], U32 argc)
{
    if (v[1].gt == GT_INT) {
        GI x = 1;

        if (v[1].i < 0) x = 0;
        for (U32 i=0; i < v[1].i; i++) {
            x *= v[0].i;;
        }
        RETURN_INT(x);
    }

#if GURU_USE_FLOAT && GURU_USE_MATH
    else if (v[1].gt == GT_FLOAT) {
        RETURN_FLOAT(pow(v[0].i, v[1].f));
    }
#endif
}


//================================================================
/*! (operator) %
 */
__GURU__ void
c_int_mod(GV v[], U32 argc)
{
    GI n = ARG_INT(1);
    RETURN_INT(v->i % n);
}

//================================================================
/*! (operator) &; bit operation AND
 */
__GURU__ void
c_int_and(GV v[], U32 argc)
{
    GI n = ARG_INT(1);
    RETURN_INT(v->i & n);
}

//================================================================
/*! (operator) |; bit operation OR
 */
__GURU__ void
c_int_or(GV v[], U32 argc)
{
    GI n = ARG_INT(1);
    RETURN_INT(v->i | n);
}

//================================================================
/*! (operator) ^; bit operation XOR
 */
__GURU__ void
c_int_xor(GV v[], U32 argc)
{
    GI n = ARG_INT(1);
    RETURN_INT(v->i ^ n);
}

//================================================================
/*! (operator) ~; bit operation NOT
 */
__GURU__ void
c_int_not(GV v[], U32 argc)
{
    GI n = ARG_INT(0);
    RETURN_INT(~n);
}

//================================================================
/*! (operator) <<; bit operation LEFT_SHIFT
 */
__GURU__ void
c_int_lshift(GV v[], U32 argc)
{
    GI n = ARG_INT(1);
    RETURN_INT(v->i << n);
}

//================================================================
/*! (operator) >>; bit operation RIGHT_SHIFT
 */
__GURU__ void
c_int_rshift(GV v[], U32 argc)
{
    GI n = ARG_INT(1);
    RETURN_INT(v->i >> n);
}

//================================================================
/*! (method) abs
 */
__GURU__ void
c_int_abs(GV v[], U32 argc)
{
    if (v[0].i < 0) {
        v[0].i = -v[0].i;
    }
}

#if GURU_USE_FLOAT
//================================================================
/*! (method) to_f
 */
__GURU__ void
c_int_to_f(GV v[], U32 argc)
{
    GF f = ARG_INT(0);
    RETURN_FLOAT(f);
}
#endif

#if GURU_USE_STRING
//================================================================
/*! (method) chr
 */
__GURU__ void
c_int_chr(GV v[], U32 argc)
{
    U8 buf[2] = { (U8)ARG_INT(0), '\0' };

    RETURN_VAL(guru_str_new(buf));
}

//================================================================
/*! (method) to_s
 */
__GURU__ void
c_int_to_s(GV v[], U32 argc)
{
	U32 i    = ARG_INT(0);
    U32 bias = 'a' - 10;
    U32 base = 10;

    if (argc) {
        base = ARG_INT(1);
        if (base < 2 || base > 36) return;	// raise ? ArgumentError
    }
    U8  buf[64+2];							// int64 + terminate + 1
    U8P p = buf + sizeof(buf) - 1;			// fill from the tail of the buffer
    U32 x;
    *p = '\0';
    do {
        x = i % base;
        *--p = (x < 10)? x + '0' : x + bias;
        x /= base;
    } while (x != 0);

    RETURN_VAL(guru_str_new(p));
}
#endif

__GURU__ void
guru_init_class_int(void)
{
    // int
    guru_class *c = guru_class_int = guru_add_class("int", guru_class_object);

    guru_add_proc(c, "[]", 		c_int_bitref);
    guru_add_proc(c, "-@", 		c_int_negative);
    guru_add_proc(c, "**", 		c_int_power);
    guru_add_proc(c, "%", 		c_int_mod);
    guru_add_proc(c, "&", 		c_int_and);
    guru_add_proc(c, "|", 		c_int_or);
    guru_add_proc(c, "^", 		c_int_xor);
    guru_add_proc(c, "~", 		c_int_not);
    guru_add_proc(c, "<<", 		c_int_lshift);
    guru_add_proc(c, ">>", 		c_int_rshift);
    guru_add_proc(c, "abs",		c_int_abs);
#if GURU_USE_FLOAT
    guru_add_proc(c, "to_f",	c_int_to_f);
#endif
#if GURU_USE_STRING
    guru_add_proc(c, "chr", 	c_int_chr);
    guru_add_proc(c, "inspect",	c_int_to_s);
    guru_add_proc(c, "to_s", 	c_int_to_s);
#endif
}

// Float
#if GURU_USE_FLOAT
//================================================================
/*! (operator) unary -
 */
__GURU__ void
c_float_negative(GV v[], U32 argc)
{
    GF f = ARG_FLOAT(0);
    RETURN_FLOAT(-f);
}

#if GURU_USE_MATH
//================================================================
/*! (operator) ** power
 */
__GURU__ void
c_float_power(GV v[], U32 argc)
{
    GF n = 0;
    switch (v[1].gt) {
    case GT_INT: 	n = v[1].i;	break;
    case GT_FLOAT:	n = v[1].d;	break;
    default: break;
    }

    RETURN_FLOAT(pow(v[0].d, n));
}
#endif

//================================================================
/*! (method) abs
 */
__GURU__ void
c_float_abs(GV v[], U32 argc)
{
    if (v[0].f < 0) {
        v[0].f = -v[0].f;
    }
}

//================================================================
/*! (method) to_i
 */
__GURU__ void
c_float_to_i(GV v[], U32 argc)
{
    GI i = (GI)ARG_FLOAT(0);
    RETURN_INT(i);
}

#if GURU_USE_STRING
//================================================================
/*! (method) to_s
 */
__GURU__ void
c_float_to_s(GV v[], U32 argc)
{
	guru_na("float#to_s");
}
#endif

//================================================================
/*! initialize class Float
 */
__GURU__ void
guru_init_class_float(void)
{
    // Float
    guru_class *c = guru_class_float = guru_add_class("Float", guru_class_object);

    guru_add_proc(c, "-@", 		c_float_negative);
#if GURU_USE_MATH
    guru_add_proc(c, "**", 		c_float_power);
#endif
    guru_add_proc(c, "abs", 	c_float_abs);
    guru_add_proc(c, "to_i", 	c_float_to_i);
#if GURU_USE_STRING
    guru_add_proc(c, "inspect", c_float_to_s);
    guru_add_proc(c, "to_s", 	c_float_to_s);
#endif
}

#endif
