/*! @file
  @brief
  Guru value definitions non-optimized

  <pre>
  Copyright (C) 2019- GreenII
  </pre>
*/
#include <stdio.h>
#include "guru.h"
#include "gurux.h"
#include "mmu.h"				// guru_malloc
#include "vmx.h"
#include "debug.h"

// forward declaration for implementation
extern "C" __GPU__  void guru_mmu_init(void *ptr, U32 sz);
extern "C" __GPU__  void guru_core_init(void);
extern "C" __GPU__  void guru_console_init(U8 *buf, U32 sz);

U8 *guru_host_heap;				// guru global memory
U8 *_guru_out;					// guru output stream
guru_ses *_ses_list = NULL; 	// session linked-list

#if GURU_CXX_CODEBASE
VM_Pool  *_vm_pool  = NULL;
#endif // GURU_CXX_CODEBASE

//
// _fetch_bytecode:
//     	read raw bytecode from input file (or stream) into CUDA managed memory
//		for later CUDA IREP image building
//
__HOST__ char *
_fetch_bytecode(const char *rite_fname)
{
  FILE *fp = fopen(rite_fname, "rb");

  if (!fp) {
    fprintf(stderr, "File not found\n");
    return NULL;
  }

  // get filesize
  fseek(fp, 0, SEEK_END);
  size_t sz = ftell(fp);
  fseek(fp, 0, SEEK_SET);

  char *req = (char*)cuda_malloc(sz, 1);			// allocate bytecode storage

  if (req) {
	  fread(req, sizeof(char), sz, fp);
  }
  fclose(fp);

  return req;
}

__HOST__ int
guru_setup(int step, int trace)
{
	hipDeviceReset();

	debug_init(trace);												// initialize logger
	debug_log("guru initializing...");

	U8 *mem = guru_host_heap = (U8*)cuda_malloc(BLOCK_MEMORY_SIZE, 1);	// allocate main block (i.e. RAM)
	if (!mem) {
		fprintf(stderr, "ERROR: failed to allocate device main memory block!\n");
		return -1;
	}
	U8 *out = _guru_out = (U8*)cuda_malloc(MAX_BUFFER_SIZE, 1);		// allocate output buffer
	if (!_guru_out) {
		fprintf(stderr, "ERROR: output buffer allocation error!\n");
		return -2;
	}
#if GURU_CXX_CODEBASE
	_vm_pool = new VM_Pool(step);
	if (!_vm_pool) {
		fprintf(stderr, "ERROR: VM memory block allocation error!\n");
		return -3;
	}
#else
	if (vm_pool_init(step)) {										// allocate VM pool
		fprintf(stderr, "ERROR: VM memory block allocation error!\n");
		return -3;
	}
#endif // GURU_CXX_CODEBASE
	_ses_list = NULL;

	guru_mmu_init<<<1,1>>>(mem, BLOCK_MEMORY_SIZE);			// setup memory management
	guru_core_init<<<1,1>>>();								// setup basic classes	(TODO: => ROM)
#if GURU_USE_CONSOLE
	guru_console_init<<<1,1>>>(out, MAX_BUFFER_SIZE);		// initialize output buffer
#endif

    U32 sz0, sz1;
	hipDeviceGetLimit((size_t *)&sz0, hipLimitStackSize);
	hipDeviceSetLimit(hipLimitStackSize, (size_t)sz0*4);
	hipDeviceGetLimit((size_t *)&sz1, hipLimitStackSize);

	debug_log("guru initialized, ready to go...");

	return 0;
}

__HOST__ int
guru_load(char *rite_name)
{
	debug_log("guru loading RITE image into ses->stdin memory...");


	guru_ses *ses = (guru_ses *)malloc(sizeof(guru_ses));
	if (!ses) return -1;		// memory allocation error

	ses->stdout = _guru_out;

	char *ins = _fetch_bytecode(rite_name);
	if (!ins) {
		fprintf(stderr, "ERROR: bytecode request allocation error!\n");
		return -1;
	}

#if GURU_CXX_CODEBASE
	int id = ses->id = _vm_pool->get(ins);
#else
	int id = ses->id = vm_get(ins);
#endif // GURU_CXX_CODEBASE
	cuda_free(ins);

	if (id==-1) {
		fprintf(stderr, "ERROR: bytecode parsing error!\n");
		return -1;
	}
	if (id==-2) {
		fprintf(stderr, "ERROR: No more VM available!\n");
		return -1;
	}

	ses->next = _ses_list;		// add to linked-list
	_ses_list = ses;

	return 0;
}

__HOST__ int
guru_run()
{
	debug_log("guru session starting...");
	debug_mmu_stat();

	// parse BITE code into each vm
	// TODO: work producer (enqueue)
#if GURU_CXX_CODEBASE
	_vm_pool->start();
#else
	for (guru_ses *ses=_ses_list; ses!=NULL; ses=ses->next) {
		if (vm_ready(ses->id)) {
			fprintf(stderr, "ERROR: VM state failed to go into READY state!\n");
		}
	}
	// kick up main loop until all VM are done
	vm_main_start();
#endif // GURU_CXX_CODEBASE
	debug_mmu_stat();
	debug_log("guru session completed.");

	return 0;
}

__HOST__ void
guru_teardown(int sig)
{
	hipDeviceReset();

	guru_ses *tmp, *ses = _ses_list;
	while (ses) {
		tmp = ses;
		ses = ses->next;
		free(tmp);
	}
}
