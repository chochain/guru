/*! @file
  @brief
  Guru value definitions non-optimized

  <pre>
  Copyright (C) 2019- GreenII
  </pre>
*/
#include <stdio.h>
#include "guru.h"
#include "gurux.h"
#include "mmu.h"				// guru_malloc
#include "vmx.h"
#include "debug.h"

// forward declaration for implementation
extern "C" __GPU__  void guru_mmu_init(void *ptr, U32 sz);
extern "C" __GPU__  void guru_core_init(void);
extern "C" __GPU__  void guru_console_init(U8 *buf, U32 sz);

U8 *_guru_out;					// guru output stream
guru_ses *_ses_list = NULL; 	// session linked-list
//
// _fetch_bytecode:
//     	read raw bytecode from input file (or stream) into CUDA managed memory
//		for later CUDA IREP image building
//
__HOST__ char *
_fetch_bytecode(const char *rite_fname)
{
  FILE *fp = fopen(rite_fname, "rb");

  if (!fp) {
    fprintf(stderr, "File not found\n");
    return NULL;
  }

  // get filesize
  fseek(fp, 0, SEEK_END);
  size_t sz = ftell(fp);
  fseek(fp, 0, SEEK_SET);

  char *req = (char*)cuda_malloc(sz, 1);			// allocate bytecode storage

  if (req) {
	  fread(req, sizeof(char), sz, fp);
  }
  fclose(fp);

  return req;
}

__HOST__ int
guru_setup(int step, int trace)
{
	hipDeviceReset();

	debug_init(trace);												// initialize logger
	debug_log("guru initializing...");

	U8 *mem = guru_host_heap = (U8*)cuda_malloc(GURU_HEAP_SIZE, 1);	// allocate main block (i.e. RAM)
	if (!mem) {
		fprintf(stderr, "ERROR: failed to allocate device main memory block!\n");
		return -1;
	}
	U8 *out = _guru_out = (U8*)cuda_malloc(OUTPUT_BUF_SIZE, 1);		// allocate output buffer
	if (!_guru_out) {
		fprintf(stderr, "ERROR: output buffer allocation error!\n");
		return -2;
	}
	if (vm_pool_init(step)) {										// allocate VM pool
		fprintf(stderr, "ERROR: VM memory block allocation error!\n");
		return -3;
	}
	_ses_list = NULL;

	guru_mmu_init<<<1,1>>>(mem, GURU_HEAP_SIZE);			// setup memory management
	guru_core_init<<<1,1>>>();								// setup basic classes	(TODO: => ROM)
#if GURU_USE_CONSOLE
	guru_console_init<<<1,1>>>(out, OUTPUT_BUF_SIZE);		// initialize output buffer
#endif
	GPU_SYNC();

    U32 sz0, sz1;
	hipDeviceGetLimit((size_t *)&sz0, hipLimitStackSize);
	hipDeviceSetLimit(hipLimitStackSize, (size_t)sz0*4);
	hipDeviceGetLimit((size_t *)&sz1, hipLimitStackSize);

	debug_log("guru initialized, ready to go...");

	return 0;
}

__HOST__ int
guru_load(char *rite_name)
{
	debug_log("guru loading RITE image into ses->stdin memory...");


	guru_ses *ses = (guru_ses *)malloc(sizeof(guru_ses));
	if (!ses) return -1;		// memory allocation error

	ses->stdout = _guru_out;

	char *ins = _fetch_bytecode(rite_name);
	if (!ins) {
		fprintf(stderr, "ERROR: bytecode request allocation error!\n");
		return -1;
	}

	int id = ses->id = vm_get(ins);
	cuda_free(ins);

	if (id==-1) {
		fprintf(stderr, "ERROR: bytecode parsing error!\n");
		return -1;
	}
	if (id==-2) {
		fprintf(stderr, "ERROR: No more VM available!\n");
		return -1;
	}

	ses->next = _ses_list;		// add to linked-list
	_ses_list = ses;

	return 0;
}

__HOST__ int
guru_run()
{
	debug_log("guru session starting...");
	debug_mmu_stat();

	// parse BITE code into each vm
	// TODO: work producer (enqueue)
	for (guru_ses *ses=_ses_list; ses!=NULL; ses=ses->next) {
		if (vm_ready(ses->id)) {
			fprintf(stderr, "ERROR: VM state failed to go into READY state!\n");
		}
	}
	// kick up main loop until all VM are done
	vm_main_start();

	debug_mmu_stat();
	debug_log("guru session completed.");

	return 0;
}

__HOST__ void
guru_teardown(int sig)
{
	hipDeviceReset();

	guru_ses *tmp, *ses = _ses_list;
	while (ses) {
		tmp = ses;
		ses = ses->next;
		free(tmp);
	}
}
