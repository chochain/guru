#include "hip/hip_runtime.h"
/*! @file
  @brief
  Guru value definitions non-optimized

  <pre>
  Copyright (C) 2018- Greenii
  </pre>
*/
#include <stdio.h>
#include "guru.h"
#include "console.h"
#include "load.h"

extern "C" void *guru_malloc(size_t sz, int mem_type);
extern "C" void dump_alloc_stat(void);

extern "C" __global__ void guru_init_alloc(void *ptr, unsigned int sz);	// in alloc.cu
extern "C" __global__ void guru_init_static(void);						// in vm.cu

int _alloc_session(guru_ses *ses, size_t req_sz, size_t res_sz)
{
	ses->req = (uint8_t *)guru_malloc(req_sz, 1);	// allocate bytecode storage
	ses->res = (uint8_t *)guru_malloc(res_sz, 1);	// allocate output buffer

	if (!ses->req || !ses->res) return 1;

    guru_init_console_buf<<<1,1>>>(ses->res, res_sz);

    return (hipSuccess==hipGetLastError()) ? 0 : 1;
}

int _input_bytecode(guru_ses *ses, const char *rite_fname)
{
  FILE *fp = fopen(rite_fname, "rb");

  if (fp==NULL) {
    fprintf(stderr, "File not found\n");
    return -1;
  }

  // get filesize
  fseek(fp, 0, SEEK_END);
  size_t sz = ftell(fp);
  fseek(fp, 0, SEEK_SET);

  int err = _alloc_session(ses, sz, MAX_BUFFER_SIZE);

  if (err != 0) {
	  fprintf(stderr, "session buffer allocation error: %d.\n", err);
	  return err;
  }
  else {
	  fread(ses->req, sizeof(char), sz, fp);
  }
  fclose(fp);

  return 0;
}

uint8_t *init_session(guru_ses *ses, const char *rite_fname)
{
	int rst = _input_bytecode(ses, rite_fname);

	if (rst != 0) return NULL;

	void *mem = guru_malloc(BLOCK_MEMORY_SIZE, 1);

    guru_init_alloc<<<1,1>>>(mem, BLOCK_MEMORY_SIZE);
	guru_init_static<<<1,1>>>();
	dump_alloc_stat();

	mrbc_vm *vm = (mrbc_vm *)guru_malloc(sizeof(mrbc_vm), 1);			// allocate bytecode storage

	guru_parse_bytecode<<<1,1>>>(vm, ses->req);
	dump_alloc_stat();

	return (uint8_t *)vm;
}
    
