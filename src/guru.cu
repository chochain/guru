/*! @file
  @brief
  Guru value definitions non-optimized

  <pre>
  Copyright (C) 2018- Greenii
  </pre>
*/
#include <stdio.h>
#include "gurux.h"
#include "vmx.h"
#include "alloc.h"				// guru_malloc

// forward declaration for implementation
extern "C" __GPU__ void guru_memory_init(void *ptr, U32 sz);
extern "C" __GPU__ void guru_global_init(void);
extern "C" __GPU__ void guru_class_init(void);
extern "C" __GPU__ void guru_console_init(U8 *buf, U32 sz);

U8P _guru_mem;			// guru global memory
U8P _guru_out;			// guru output stream
guru_ses *_ses_list;			// session linked-list

__HOST__ U8P
_get_request_bytecode(const U8P rite_fname)
{
  FILE *fp = fopen((const char *)rite_fname, "rb");

  if (!fp) {
    fprintf(stderr, "File not found\n");
    return NULL;
  }

  // get filesize
  fseek(fp, 0, SEEK_END);
  size_t sz = ftell(fp);
  fseek(fp, 0, SEEK_SET);

  U8P req = (U8P)guru_malloc(sz, 1);	// allocate bytecode storage

  if (req) {
	  fread(req, sizeof(char), sz, fp);
  }
  fclose(fp);

  return req;
}

__HOST__ U32
_session_add(guru_ses *ses, const U8P rite_fname, U32 trace)
{
	ses->trace = trace;
	ses->out   = _guru_out;

	U8P in = ses->in = _get_request_bytecode(rite_fname);
	if (!in) {
		fprintf(stderr, "ERROR: bytecode request allocation error!\n");
		return 3;
	}
	hipError_t rst = guru_vm_setup(ses, trace);
	if (hipSuccess != rst) {
		fprintf(stderr, "ERROR: virtual memory block allocation error!\n");
		return 1;
	}
	if (trace) guru_dump_alloc_stat();

	ses->next = _ses_list;		// add to linked-list
	_ses_list = ses;

	return rst;
}

__HOST__ U32
guru_setup(U32 trace)
{
	U8P mem = _guru_mem = (U8P)guru_malloc(BLOCK_MEMORY_SIZE, 1);
	if (!_guru_mem) {
		fprintf(stderr, "ERROR: failed to allocate device main memory block!\n");
		return 1;
	}
	U8P out = _guru_out = (U8P)guru_malloc(MAX_BUFFER_SIZE, 1);	// allocate output buffer
	if (!_guru_out) {
		fprintf(stderr, "ERROR: output buffer allocation error!\n");
		return 2;
	}
	_ses_list = NULL;

	guru_memory_init<<<1,1>>>(mem, BLOCK_MEMORY_SIZE);			// setup memory management
	guru_global_init<<<1,1>>>();								// setup static objects
	guru_class_init<<<1,1>>>();									// setup basic classes
    guru_console_init<<<1,1>>>(out, MAX_BUFFER_SIZE);			// initialize output buffer

    U32 sz0, sz1;
	hipDeviceGetLimit((size_t *)&sz0, hipLimitStackSize);
	hipDeviceSetLimit(hipLimitStackSize, (size_t)sz0*4);
	hipDeviceGetLimit((size_t *)&sz1, hipLimitStackSize);

	if (trace) {
		printf("guru session initialized[defaultStackSize %d => %d]\n", sz0, sz1);
		guru_dump_alloc_stat();
	}
	return 0;
}

__HOST__ U32
guru_load(U8 **argv, U32 n, U32 trace)
{
	guru_ses *ses = (guru_ses *)malloc(sizeof(guru_ses) * n);

	if (!ses) return 1;			// memory allocation error

	for (U32 i=1; i<=n; i++, ses++) {
		_session_add(ses, argv[i], trace);
	}
	return 0;
}

__HOST__ U32
guru_run(U32 trace)
{
	hipError_t rst = guru_vm_run(_ses_list, trace);
    if (hipSuccess != rst) {
    	fprintf(stderr, "\nERR> %s\n", hipGetErrorString(rst));
    }
	if (trace) {
		printf("guru_session completed\n");
		guru_dump_alloc_stat();
	}
	rst = guru_vm_release(_ses_list, trace);

	return hipSuccess != rst;
}


