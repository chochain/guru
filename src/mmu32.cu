#include "hip/hip_runtime.h"
/*! @file
  @brief
  GURU 32-bit memory management.

  <pre>
  Copyright (C) 2019 GreenII.

  This file is distributed under BSD 3-Clause License.

  Memory management for objects in GURU.

  </pre>
*/
#include "guru_config.h"
#include "guru.h"
#include "util.h"
#include "mmu.h"
#include "mmu32.h"

// TLSF: Two-Level Segregated Fit allocator with O(1) time complexity.
// Layer 1st(f), 2nd(s) model, smallest block 16-bytes, 16-byte alignment
// TODO: multiple-pool, thread-safe
// semaphore
#define _LOCK			{ MUTEX_LOCK(_mutex_mem); }
#define _UNLOCK			{ MUTEX_FREE(_mutex_mem); }

// memory pool
__GURU__ U8				*guru_device_heap;						// CUDA kernel global memory pool
__GURU__ U32 			_heap_size;
__GURU__ U32 			_mutex_mem;

// free memory bitmap
__GURU__ U32 			_l1_map;								// use lower 24 bits
__GURU__ U8 			_l2_map[L1_BITS];						// 8-bit, (16-bit requires too many FL_SLOTS)
__GURU__ free_block		*_free_list[FL_SLOTS];

#if GURU_DEBUG
#define MMU_CHECK		ASSERT(__mmu_ok())
//================================================================
/*! statistics

  @param  *total	returns total memory.
  @param  *used		returns used memory.
  @param  *free		returns free memory.
  @param  *fragment	returns memory fragmentation
*/
#define bin2u32(x) ((x << 24) | ((x & 0xff00) << 8) | ((x >> 8) & 0xff00) | (x >> 24))

__GURU__ void
_dump_freelist(const char *hdr, int sz)
{
	PRINTF("!!!%6s(x%04x) L1=%04x:", hdr, sz, _l1_map);
	for (int i=0; i<L1_BITS; i++) { PRINTF(" %02x", _l2_map[i]); }
	for (int i=0; i<FL_SLOTS; i++) {
		if (!_free_list[i]) continue;
		PRINTF(" [%02x]=>[", i);
		for (free_block *b = _free_list[i]; b!=NULL; b=NEXT_FREE(b)) {
			U32 a = (U32A)b;		// when using b directly, higher bit will bleed into second parameter
			PRINTF(" %06x:%04x", a & 0xffffff, b->bsz);
			if (IS_USED(b)) {
				PRINTF("<-USED?");
				break;				// something is wrong (link is broken here)
			}
		}
		PRINTF(" ] ");
	}
	PRINTF("\n");
}

__GPU__ void
_mmu_freelist()
{
	if (threadIdx.x!=0 || blockIdx.x!=0) return;

	_dump_freelist("check", 0);
}

//================================================================
// MMU JTAG sanity check - memory pool walker
//
__GURU__ int
__mmu_ok()											// mmu sanity check
{
	used_block *p0 = (used_block*)guru_device_heap;
	used_block *p1 = (used_block*)BLK_AFTER(p0);
	U32 tot = sizeof(free_block);
	while (p1) {
		if (p0->bsz != (p1->psz&~FREE_FLAG)) {		// ERROR!
			return 0;								// memory integrity broken!
		}
		tot += p0->bsz;
		p0  = p1;
		p1  = (used_block*)BLK_AFTER(p0);
	}
#if CC_DEBUG
	if (tot!=_heap_size) {							// ERROR, tally off
		return 0;									// debug break point
	}
#endif // CC_DEBUG
	return (tot==_heap_size && !p1);				// last check
}

__GPU__ void
_alloc_stat(guru_mstat *s)
{
	if (threadIdx.x!=0 || blockIdx.x!=0) return;

	guru_mstat v;
	guru_mmu_stat(&v);

	*s = v;
}

__HOST__ U32
guru_mmu_check(U32 level)
{
	if (level==0) return 0;
	if (level & 1) {
		guru_mstat *s;
		hipMallocManaged(&s, sizeof(guru_mstat));				// allocate host memory

		_alloc_stat<<<1,1>>>(s);
		GPU_SYNC();

		printf("%14smem=%d(0x%x): free=%d(0x%x), used=%d(0x%x), nblk=%d, nfrag=%d, %d%% allocated\n",
			"", s->total, s->total, s->free, s->free, s->used, s->used, s->nblk, s->nfrag, s->pct_used);
		hipFree(s);
	}
	if (level & 2) {
		_mmu_freelist<<<1,1>>>();
		GPU_SYNC();
	}
	return 0;
}
#else
__HOST__ U32 guru_mmu_check(U32 trace);
#define MMU_CHECK
#endif // GURU_DEBUG

//================================================================
// most significant bit that is set
// __xls(i) = 32-__ffs(__brev(i))
//
__GURU__ __INLINE__ U32
__xls(U32 x)
{
	U32 n;
	asm("bfind.u32 %0, %1;\n\t" : "=r"(n) : "r"(x));
	return n;
}
// least significant bit that is set
// __xfs(i) = __ffs(i)+1;
//
__GURU__ __INLINE__ U32
__xfs(U32 x)
{
	U32 n;
	asm(
		"brev.b32 %0, %1;\n\t"
		"clz.b32 %0, %0;\n\t"
		: "=r"(n) : "r"(x)
	);
	return n;
}
//================================================================
/*! calc f and s, and returns fli,sli of free_blocks

  @param  alloc_size	alloc size
  @retval int			index of free_blocks
*/
__GURU__ U32
__idx(U32 sz)
{
/* old
	U32 v  = __xls(sz);
	U32 l1 = v<BASE_BITS ? 0 : v - BASE_BITS + 1;	// 1st level index

	U32 n  = l1<2 ? 0 : l1 - 1;						// down shifting bit
    U32 l2 = (sz >> (n+MN_BITS)) & L2_MASK; 		// 2nd level index (with lower bits)
*/
    U32 v  = __xls(sz) + 1;							// 1 means LSB
    U32 l1 = v > BASE_BITS ? v - BASE_BITS : 0;
    U32 n  = v > BASE_BITS ? v - MN_BITS : MN_BITS;
    U32 l2 = sz >> n;
//    PRINTF("!!!sz=%04x:v=%1x, L1=%02x,L2=%02x => INDEX=%x\n", sz, v, l1, l2, INDEX(l1, l2));
    return INDEX(l1, l2);
}

//================================================================
/*! wipe the free_block from linked list

  @param  blk	pointer to free block.
*/
__GURU__ void
__unmap(free_block *blk)
{
	ASSERT(IS_FREE(blk));						// ensure block is free

	U32 index = __idx(blk->bsz);
    free_block *n = _free_list[index] = NEXT_FREE(blk);
    if (n) {									// up link
    	// blk->next->prev = blk->prev;
    	n->prev = blk->prev ? U8POFF(n, PREV_FREE(blk)) : 0;
    	ASSERT((n->prev&7)==0);
    }
    else {										// 1st of the link
        CLEAR_MAP(index);						// clear the index bit
    }
    if (blk->prev) {							// down link
    	free_block *p = PREV_FREE(blk);
    	// blk->prev->next = blk->next;
    	p->next = blk->next ? U8POFF(n, p) : 0;
    }
    blk->next = blk->prev = 0xeeeeeeee;			// wipe for debugging

    MMU_CHECK;
}

//================================================================
/*! merge p0 and p1 adjacent free blocks.
  ptr2 will disappear

  @param  ptr1	pointer to free block 1
  @param  ptr2	pointer to free block 2
*/
__GURU__ void
__pack(free_block *b0, free_block *b1)
{
	ASSERT((free_block*)BLK_AFTER(b0)==b1);
	ASSERT(IS_FREE(b1));

	// remove b0, b1 from free list first (sizes will not change)
    __unmap(b1);

	// merge b0 and b1, retain b0.FREE_FLAG
	used_block *b2 = (used_block *)BLK_AFTER(b1);
	b2->psz += b1->psz & ~FREE_FLAG;	// watch for the block->flag
    b0->bsz += b1->bsz;					// include the block header

#if GURU_DEBUG
    *((U64*)b1) = 0xeeeeeeeeeeeeeeee;	// wipe b1 header
#endif
    MMU_CHECK;
}

//================================================================
/*! Mark that block free and register it in the free index table.

  @param  blk	Pointer to block to be freed.

  TODO: check thread safety
*/
__GURU__ void
_mark_free(free_block *blk)
{
	ASSERT(IS_USED(blk));

	U32 index = __idx(blk->bsz);
#if CC_DEBUG
	U32 *l1m = &_l1_map;
	U8  *l2m = _l2_map;
    U32 l1 = L1(index);
    U32 l2 = L2(index);
    U32 t1 = TIC(l1);
    U32 t2 = TIC(l2);
    U32 m1 = L1_MAP(index);
    U32 m2 = L2_MAP(index);
    U32 x  = *l1m;
#endif // CC_DEBUG

    SET_MAP(index);								// set ticks for available maps

#if CC_DEBUG
    U32 m1x = L1_MAP(index);
    U32 m2x = L2_MAP(index);
#endif // CC_DEBUG
    // update block attributes
    free_block *head = _free_list[index];

    ASSERT(head!=blk);

    SET_FREE(blk);
    blk->next = head ? U8POFF(head, blk) : 0;	// setup linked list
    ASSERT((blk->next&7)==0);
    blk->prev = 0;
    if (head) {									// non-end block, add backward link
    	head->prev = U8POFF(blk, head);
        ASSERT((head->prev&7)==0);
    	SET_FREE(head);							// turn the free flag back on
    }
    _free_list[index] = blk;					// new head of the linked list
}

__GURU__ free_block*
_mark_used(U32 index)
{
    free_block *blk  = _free_list[index];
    ASSERT(blk);
    ASSERT(IS_FREE(blk));

    __unmap(blk);
    SET_USED(blk);

    return blk;
}

__GURU__ void
_merge_with_next(free_block *b0)
{
	free_block *b1 = (free_block *)BLK_AFTER(b0);
	while (b1 && IS_FREE(b1) && b1->bsz!=0) {
		__pack(b0, b1);
		b1 = (free_block *)BLK_AFTER(b0);	// try the already expanded block again
	}
}

__GURU__ free_block*
_merge_with_prev(free_block *b1)
{
    free_block *b0 = (free_block *)BLK_BEFORE(b1);
	if (b0==NULL || IS_USED(b0)) return b1;

	__unmap(b0);							// take it out of free_list before merge
	__pack(b0, b1);							// take b1 out and merge with b0

	SET_USED(b0);							// _mark_free assume b0 to be a USED block
	_mark_free(b0);

    return b0;
}

//================================================================
/*! Find index to a free block

  @param  size	size
  @retval -1	not found
  @retval index to available _free_list
*/
__GURU__ S32
_find_free_index(U32 sz)
{
    U32 index = __idx(sz);						// find free_list index by size

    if (_free_list[index]) return index;		// free block available, use it

    // no previous block exist, create a new one
    U32 l1  = L1(index);
    U32 l2  = L2(index);
    U32 avl = _l2_map[l1];			    		// check any 2nd level available
    if (avl >> l2) {
    	l2 = __xls(avl);						// get first available l2 index
    }
    else if ((avl = _l1_map)) {					// check if 1st level available
        l1 = __xls(avl);        				// allocate new 1st & 2nd level indices
        l2 = __xls(_l2_map[l1]);
    }
    else return -1;								// out of memory

    return INDEX(l1, l2);               		// index to freelist head
}

//================================================================
/*! Split free block by size (before allocating)

  @param  blk	pointer to free block
  @param  size	storage size
*/
__GURU__ void
_split(free_block *blk, U32 bsz)
{
	ASSERT(IS_USED(blk));

    if ((bsz + MIN_BLOCK + sizeof(free_block)) > blk->bsz) return;	// too small to split


    // split block, free
    free_block *free = (free_block *)U8PADD(blk, bsz);				// future next block (i.e. alot bsz bytes)
    free_block *aft  = (free_block *)BLK_AFTER(blk);				// next adjacent block

    free->bsz = blk->bsz - bsz;										// carve out the acquired block
    free->psz = U8POFF(free, blk);									// positive offset to previous block
    blk->bsz  = bsz;												// allocate target block

    if (aft) {
        aft->psz = U8POFF(aft, free)|(aft->psz&FREE_FLAG);			// backward offset (positive)
        _merge_with_next(free);										// _combine if possible
    }
    _mark_free(free);			// add to free_list and set (free, tail, next, prev) fields

    MMU_CHECK;
}

//================================================================
/*! initialize

  @param  ptr	pointer to free memory block.
  @param  size	size. (max 4G)
*/
__GURU__ void
_init_mmu(void *mem, U32 heap_size)
{
    ASSERT(heap_size > 0);

    U32 bsz = heap_size - sizeof(free_block);

    guru_device_heap = (U8*)mem;
    _heap_size   = heap_size;
    _mutex_mem	 = 0;

    // initialize entire memory pool as the first block
    free_block *head  = (free_block*)guru_device_heap;
    head->bsz = bsz;						// 1st (big) block
    head->psz = 0;
    SET_USED(head);

    _mark_free(head);						// will set free, tail, next, prev

    free_block *tail = (free_block*)BLK_AFTER(head);	// last block
    tail->bsz = tail->next = tail->prev = 0;
    tail->psz = bsz;
    SET_USED(tail);

    MMU_CHECK;
}

//================================================================
/*! allocate memory

  @param  size	request storage size.
  @return void* pointer to a guru memory block.
*/
__GURU__ void*
guru_alloc(U32 sz)
{
	if (sz < 4) {
		sz += 1; sz -= 1;
	}
    U32 bsz = sz + sizeof(used_block);			// logical => physical size
    CHECK_MEMSZ(bsz);							// check alignment & sizing

    _LOCK;
	U32 index 		= _find_free_index(bsz);
	free_block *blk = _mark_used(index);		// take the indexed block off free list

	_split(blk, bsz);							// allocate the block, free up the rest
	_UNLOCK;

#if GURU_DEBUG
    _dump_freelist("alloc", sz);
    U32 *p = (U32*)BLK_DATA(blk);				// point to raw space allocated
    sz >>= 2;
    for (int i=0; i < (sz>16 ? 16 : sz); i++) *p++ = 0xaaaaaaaa;
#endif

	return BLK_DATA(blk);						// pointer to raw space
}

//================================================================
/*! re-allocate memory

  @param  ptr	Return value of raw malloc()
  @param  size	request size
  @return void* pointer to allocated memory.
*/
__GURU__ void*
guru_realloc(void *p0, U32 sz)
{
	ASSERT(p0);

	U32 bsz = sz + sizeof(used_block);					// include the header
	CHECK_MEMSZ(bsz);									// assume it is aligned already

    used_block *blk = (used_block *)BLK_HEAD(p0);
    ASSERT(IS_USED(blk));								// make sure it is used

    if (bsz > blk->bsz) {
    	_merge_with_next((free_block *)blk);			// try to get the block bigger
    }
    if (bsz == blk->bsz) return p0;						// fits right in
    if (bsz < blk->bsz) {								// enough space now
    	if ((blk->bsz - bsz) > (sizeof(used_block)+MIN_BLOCK)) {	// but is it too big?
    		_split((free_block*)blk, bsz);				// allocate the block, free up the rest
    	}
    	return p0;
    }
    // not big enough block found, new alloc and deep copy
    void *p1 = guru_alloc(bsz);
    MEMCPY(p1, p0, sz);									// deep copy, !!using CUDA provided memcpy

    guru_free(p0);										// reclaim block
#if GURU_DEBUG
	_dump_freelist("ralloc", sz);
#endif // GURU_DEBUG

    return p1;
}

__GURU__ GR*
guru_gr_alloc(U32 n)
{
	return (GR*)guru_alloc(sizeof(GR) * n);
}

__GURU__ GR*
guru_gr_realloc(GR *gv, U32 n)
{
	return (GR*)guru_realloc(gv, sizeof(GR) * n);
}

//================================================================
/*! release memory
*/
__GURU__ void
guru_free(void *ptr)
{
	if (!ptr) return;

	_LOCK;
    free_block *blk = (free_block *)BLK_HEAD(ptr);			// get block header
    U32 sz = blk->bsz;

    _merge_with_next(blk);
#if GURU_DEBUG
    if (BLK_AFTER(blk)) {
    	U32 *p = (U32*)U8PADD(blk, sizeof(used_block));
    	U32 sz = blk->bsz ? (blk->bsz - sizeof(used_block))>>2 : 0;
    	for (int i=0; i< (sz>32 ? 32 : sz); i++) *p++=0xffffffff;
    }
#endif
    _mark_free(blk);

    // the block is free now, try to merge a free block before if exists
    blk = _merge_with_prev(blk);
    _UNLOCK;

    MMU_CHECK;
#if GURU_DEBUG
	_dump_freelist("free", sz);
#endif // GURU_DEBUG
}

//================================================================
/*! release memory, vm used.

  @param  vm	pointer to VM.
*/
__GURU__ void
guru_mmu_clr()
{
	used_block *p = (used_block *)guru_device_heap;
    while (p) {
    	if (IS_USED(p)) {
    		guru_free(BLK_DATA(p));		// pointer to raw space
    	}
    	p = (used_block *)BLK_AFTER(p);
    }
}

__GURU__ void
guru_mmu_stat(guru_mstat *s)
{
	used_block *p = (used_block *)guru_device_heap;
	MEMSET(s, 0, sizeof(guru_mstat));	// wipe, !using CUDA provided memset

	U32 flag = IS_FREE(p);				// starting block type
	while (p) {							// walk the memory pool
		U32 bsz = p->bsz;				// current block size
		if (flag != IS_FREE(p)) {       // supposed to be merged
			s->nfrag++;
			flag = IS_FREE(p);
		}
		s->total += bsz;
		s->nblk  += 1;
		if (IS_FREE(p)) {
			s->nfree += 1;
			s->free  += bsz;
		}
		else {
			s->nused += 1;
			s->used  += bsz;
		}
		p = (used_block *)BLK_AFTER(p);
	}
	s->total    += sizeof(free_block);
	s->pct_used = (int)(100*(s->used+1)/s->total);
}

__GPU__ void
guru_mmu_init(void *ptr, U32 sz)
{
	if (threadIdx.x!=0 || blockIdx.x!=0) return;

	_init_mmu(ptr, sz);
}


__HOST__ void*
cuda_malloc(U32 sz, U32 type)
{
	void *mem;

	// TODO: to add texture memory
	switch (type) {
	case 0: 	hipMalloc(&mem, sz); break;			// allocate device memory
	default: 	hipMallocManaged(&mem, sz);			// managed (i.e. paged) memory
	}
    if (hipSuccess != hipGetLastError()) return NULL;

    return mem;
}

__HOST__ void
cuda_free(void *mem) {
	hipFree(mem);
}

