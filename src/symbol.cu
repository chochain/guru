#include "hip/hip_runtime.h"
/*! @file
  @brief
  GURU Symbol class (implemented as a string hasher)

  <pre>
  Copyright (C) 2019- GreenII

  This file is distributed under BSD 3-Clause License.

  </pre>
*/
#include <assert.h>

#include "value.h"
#include "mmu.h"
#include "class.h"
#include "symbol.h"
#include "c_string.h"
#include "c_array.h"
#include "inspect.h"

__GURU__ U32 	_sym_idx = 0;					// point to the last(free) sym_list array.
__GURU__ U8*	_sym[MAX_SYMBOL_COUNT];
__GURU__ U32	_sym_hash[MAX_SYMBOL_COUNT];

//================================================================
/*! Calculate hash value.

  @param  str		Target string.
  @return uint16_t	Hash value.
*/
__GURU__ U32
_calc_hash(const U8 *str)
{
    U32 h = 0;
    for (U32 i=0, b=STRLENB(str); i<b; i++) {
        h = h * 37 + *str++;		// a simplistic hashing algo
    }
    return h;
}

//================================================================
/*! search index table
 */
__GPU__ void
__scan(S32 *idx, const U32 hash)
{
	S32 i = threadIdx.x;

	if (i<_sym_idx && _sym_hash[i]==hash) *idx = i;

	__syncthreads();
}

__GURU__ S32
_search_index(const U8 *str)
{
	U32 hash = _calc_hash(str);

	static S32 idx;					// warn: scoped outside of this function

	idx = -1;
    __scan<<<1, 32*(1+(_sym_idx>>5))>>>(&idx, hash);
	hipDeviceSynchronize();

    return idx;
}

//================================================================
/*! add to index table (assume no entry exists)
 */
__GURU__ U32
_add_index(const U8 *str)
{
    // append table.
    U32 idx  = _sym_idx++;
    assert(idx<MAX_SYMBOL_COUNT);

    // deep copy the string (can shallow work?)
    U32 asz  = STRLENB(str) + 1;	ALIGN(asz);
    U8  *buf = (U8*)guru_alloc(asz);

    MEMCPY(buf, str, asz);
    _sym[idx]      = (U8*)buf;
    _sym_hash[idx] = _calc_hash(str);

    return idx;
}

//================================================================
/*! Convert string to symbol value.

  @param  str		Target string.
  @return GS	Symbol value.
*/
__GURU__ GS
name2id(const U8 *str)
{
    S32 sid = _search_index(str);

    if (sid<0) {    // create new symbol entry
    	sid = _add_index(str);
#if CC_DEBUG
        printf("\sym[%2d]%08x=>%s\n", sid, _sym_hash[sid], _sym[sid]);
    }
    else {
    	printf("\sym[%2d]%08x: %s\n", sid, _sym_hash[sid], _sym[sid]);
#endif // CC_DEBUG
    }
    return sid;
}

//================================================================
/*! Convert symbol value to string.

  @param  GS	Symbol value.
  @return const char*	String.
  @retval NULL		Invalid sym_id was given.
*/
__GURU__ U8*
id2name(GS sid)
{
    return (sid < _sym_idx) ? _sym[sid] : NULL;
}

//================================================================
/*! constructor

  @param  vm	pointer to VM.
  @param  str	String
  @return 	symbol object
*/
__GURU__ GV
guru_sym_new(const U8 *str)
{
    GV v; { v.gt = GT_SYM; v.acl=0; v.i=name2id(str); }

    return v;
}

//================================================================
// call by symbol
#if !GURU_USE_ARRAY
__CFUNC__	sym_all(GV v[], U32 vi)	{}
#else
__CFUNC__
sym_all(GV v[], U32 vi)
{
    GV ret = guru_array_new(_sym_idx);

    for (U32 i=0; i < _sym_idx; i++) {
        GV sym1; { sym1.gt = GT_SYM; sym1.acl=0; sym1.i=1; }
        guru_array_push(&ret, &sym1);
    }
    RETURN_VAL(ret);
}
#endif // GURU_USE_ARRAY

__CFUNC__
sym_to_s(GV v[], U32 vi)
{
	GV ret = guru_str_new(id2name(v->i));
    RETURN_VAL(ret);
}

__CFUNC__ sym_nop(GV v[], U32 vi) {	/* do nothing */	}

//================================================================
/*! initialize
 */
__GURU__ __const__ Vfunc sym_vtbl[] = {
	{ "id2name", 	gv_to_s		},
	{ "to_sym",     sym_nop		},
	{ "to_s", 		sym_to_s	}, 	// no leading ':'
	{ "inspect", 	gv_to_s		},
	{ "all_symbols", sym_all	}
};

__GURU__ void
guru_init_class_symbol()  // << from symbol.cu
{
    guru_rom_set_class(GT_SYM, "Symbol", GT_OBJ, sym_vtbl, VFSZ(sym_vtbl));
}

__GPU__ void
_id2str(GS sid, U8 *str)
{
	if (blockIdx.x!=0 || threadIdx.x!=0) return;

	U8 *s = id2name(sid);
	STRCPY(str, s);
}

#if GURU_DEBUG
__HOST__ void
id2name_host(GS sid, U8 *str)
{
	_id2str<<<1,1>>>(sid, str);
	DEVSYNC();
}
#endif // GURU_DEBUG
