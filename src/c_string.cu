#include "hip/hip_runtime.h"
/*! @file
  @brief
  mruby/c String object

  <pre>
  Copyright (C) 2015-2018 Kyushu Institute of Technology.
  Copyright (C) 2015-2018 Shimane IT Open-Innovation Center.

  This file is distributed under BSD 3-Clause License.

  </pre>
*/
#include <stdio.h>
#include <assert.h>

#include "value.h"
#include "alloc.h"
#include "static.h"
#include "symbol.h"
#include "c_string.h"

#include "puts.h"

#if GURU_USE_STRING
//================================================================
/*! white space character test

  @param  ch	character code.
  @return	result.
*/
__GURU__ bool
_is_space(U8 ch)
{
    static const char ws[] = " \t\r\n\f\v";	// '\0' on tail

    for (U32 i=0; i < sizeof(ws); i++) {
        if (ch==ws[i]) return true;
    }
    return false;
}

//================================================================
/*! get size
 */
__GURU__ __INLINE__ U32
_size(const mrbc_value *v)
{
    return v->str->len;
}

//================================================================
/*! get c-language string (U8P)
 */
__GURU__ __INLINE__ U8P
_data(const mrbc_value *v)
{
    return (U8P)v->str->data;
}

//================================================================
/*! constructor

  @param  vm	pointer to VM.
  @param  src	source string or NULL
  @param  len	source length
  @return 	string object
*/
__GURU__ mrbc_value
_new(const U8P src, U32 len)
{
    mrbc_value ret = {.tt = GURU_TT_STRING};
    /*
      Allocate handle and string buffer.
    */
    guru_str *h = (guru_str *)mrbc_alloc(sizeof(guru_str));

    assert(h!=NULL);			// out of memory
#if GURU_64BIT_ALIGN_REQUIERD
    assert(((U32A)h & 7)==0);
#endif

    U8P s = (U8P)mrbc_alloc(len);
    if (s==NULL) {					// ENOMEM
        mrbc_free(h);
        return ret;
    }
#if GURU_64BIT_ALIGN_REQUIRED
    assert(((U32A)s & 7)==0);
#endif

    // deep copy source string
    if (src==NULL) 	s[0] = '\0';
    else 			MEMCPY(s, src, len+1);		// plus '\0'

    h->refc = 1;
    h->tt   = GURU_TT_STRING;	// TODO: for DEBUG
    h->len  = len;
    h->data = s;

    ret.str = h;

    return ret;
}

//================================================================
/*! duplicate string

  @param  vm	pointer to VM.
  @param  s1	pointer to target value 1
  @param  s2	pointer to target value 2
  @return	new string as s1 + s2
*/
__GURU__ mrbc_value
_dup(const mrbc_value *v0)
{
    guru_str *h0 = v0->str;

    mrbc_value v1 = _new(NULL, h0->len);		// refc already set to 1
    if (v1.str==NULL) return v1;				// ENOMEM

    MEMCPY(v1.str->data, h0->data, h0->len + 1);

    return v1;
}

//================================================================
/*! locate a substring in a string

  @param  src		pointer to target string
  @param  pattern	pointer to substring
  @param  offset	search offset
  @return		position index. or minus value if not found.
*/
__GURU__ S32
_index(const mrbc_value *v, const mrbc_value *pattern, U32 offset)
{
    U8P p0 = _data(v) + offset;
    U8P p1 = _data(pattern);
    S32 try_cnt = _size(v) - _size(pattern) - offset;

    while (try_cnt >= 0) {
        if (MEMCMP(p0, p1, _size(pattern))==0) {
            return p1 - _data(v);	// matched.
        }
        try_cnt--;
        p0++;
    }
    return -1;
}

//================================================================
/*! remove the whitespace in myself

  @param  src	pointer to target value
  @param  mode	1:left-side, 2:right-side, 3:each
  @return	0 when not removed.
*/
__GURU__ U32
_strip(mrbc_value *v, U32 mode)
{
    U8P p0 = _data(v);
    U8P p1 = p0 + _size(v) - 1;

    // left-side
    if (mode & 0x01) {
        while (p0 <= p1) {
            if (*p0=='\0') break;
            if (!_is_space(*p0)) break;
            p0++;
        }
    }
    // right-side
    if (mode & 0x02) {
        while (p0 <= p1) {
            if (!_is_space(*p1)) break;
            p1--;
        }
    }
    U32 new_len = p1 - p0 + 1;
    if (_size(v)==new_len) return 0;

    U8P buf = _data(v);
    if (p0 != buf) MEMCPY(buf, p0, new_len);
    buf[new_len] = '\0';

    v->str->data = (U8P)mrbc_realloc(buf, new_len+1);	// shrink suitable size.
    v->str->len = new_len;

    return 1;
}

//================================================================
/*! remove the CR,LF in myself

  @param  src	pointer to target value
  @return	0 when not removed.
*/
__GURU__ int
_chomp(mrbc_value *v)
{
    U8P p0 = _data(v);
    U8P p1 = p0 + _size(v) - 1;

    if (*p1=='\n') p1--;
    if (*p1=='\r') p1--;

    U32 new_len = p1 - p0 + 1;
    if (_size(v)==new_len) return 0;

    U8P buf = _data(v);
    buf[new_len] = '\0';
    v->str->len = new_len;

    return 1;
}

//================================================================
/*! constructor by c string

  @param  vm	pointer to VM.
  @param  src	source string or NULL
  @return 	string object
*/
__GURU__ mrbc_value
guru_str_new(const U8 *src)			// cannot use U8P, need lots of casting
{
    return _new((U8P)src, STRLEN((U8P)src));
}

//================================================================
/*! destructor

  @param  str	pointer to target value
*/
__GURU__ void
guru_str_delete(mrbc_value *v)
{
    mrbc_free(v->str->data);
    mrbc_free(v->str);
}

//================================================================
/*! append string (s1 += s2)

  @param  s1	pointer to target value 1
  @param  s2	pointer to target value 2
  @param	mrbc_error_code
*/
__GURU__ void
guru_str_append(const mrbc_value *v0, const mrbc_value *v1)
{
    U32 len0 = v0->str->len;
    U32 len1 = (v1->tt==GURU_TT_STRING) ? v1->str->len : 1;

    U8P s = (U8P)mrbc_realloc(v0->str->data, len0+len1+1);		// +'\0'

    assert(s!=NULL);						// out of memory
#if GURU_64BIT_ALIGN_REQUIRED
    assert(((U32A)s & 7)==0);
#endif
    if (v1->tt==GURU_TT_STRING) {			// append str2
        MEMCPY(s + len0, v1->str->data, len1 + 1);
    }
    else if (v1->tt==GURU_TT_FIXNUM) {
        s[len0]   = v1->i;
        s[len0+1] = '\0';
    }
    v0->str->len  = len0 + len1;
    v0->str->data = s;
}

//================================================================
/*! append c string (s1 += s2)

  @param  s1	pointer to target value 1
  @param  s2	pointer to char (c_str)
  @param	mrbc_error_code
*/
__GURU__ void
guru_str_append_cstr(const mrbc_value *v0, const U8 *str)
{
    U32 len0 = v0->str->len;
    U32 len1 = STRLEN(str);

    U8P buf  = (U8P)mrbc_realloc(v0->str->data, len0+len1+1);

    assert(buf!=NULL);						// out of memory
#if GURU_64BIT_ALIGN_REQUIRED
    assert(((U32A)buf & 7)==0);
#endif
    MEMCPY(buf + len0, v0, len1 + 1);

    v0->str->len  = len0 + len1;
    v0->str->data = buf;
}

//================================================================
/*! add string (s1 + s2)

  @param  vm	pointer to VM.
  @param  s1	pointer to target value 1
  @param  s2	pointer to target value 2
  @return	new string as s1 + s2
*/
__GURU__ mrbc_value
guru_str_add(const mrbc_value *v0, const mrbc_value *v1)
{
    guru_str *h0 = v0->str;
    guru_str *h1 = v1->str;

    mrbc_value  v  = _new(NULL, h0->len + h1->len);
    guru_str *s = v.str;

    MEMCPY(s->data,           h0->data, h0->len);
    MEMCPY(s->data + h0->len, h1->data, h1->len + 1);	// include the '\0'

    return v;
}

//================================================================
/*! (method) +
 */
__GURU__ void
c_string_add(mrbc_value v[], U32 argc)
{
    if (v[1].tt != GURU_TT_STRING) {
        guru_na("str + other type");
    }
    else {
    	SET_RETURN(guru_str_add(v, v+1));
    }
}

//================================================================
/*! (method) *
 */
__GURU__ void
c_string_mul(mrbc_value v[], U32 argc)
{
    if (v[1].tt != GURU_TT_FIXNUM) {
        PRINTF("TypeError\n");	// raise?
        return;
    }
    mrbc_value ret = _new(NULL, _size(v) * v[1].i);
    if (ret.str==NULL) return;		// ENOMEM

    U8P p = (U8P)ret.str->data;
    for (U32 i = 0; i < v[1].i; i++) {
        MEMCPY(p, (U8P)_data(v), _size(v));
        p += _size(v);
    }
    *p = '\0';

    SET_RETURN(ret);
}

//================================================================
/*! (method) size, length
 */
__GURU__ void
c_string_size(mrbc_value v[], U32 argc)
{
    guru_int size = _size(v);

    SET_INT_RETURN(size);
}

//================================================================
/*! (method) to_i
 */
__GURU__ void
c_string_to_i(mrbc_value v[], U32 argc)
{
    U32 base = 10;
    if (argc) {
        base = v[1].i;
        if (base < 2 || base > 36) return;	// raise ? ArgumentError
    }
    guru_int i = guru_atoi(_data(v), base);

    SET_INT_RETURN(i);
}

#if GURU_USE_FLOAT
//================================================================
/*! (method) to_f
 */
__GURU__ void
c_string_to_f(mrbc_value v[], U32 argc)
{
    guru_float d = ATOF(_data(v));

    SET_FLOAT_RETURN(d);
}
#endif

//================================================================
/*! (method) <<
 */
__GURU__ void
c_string_append(mrbc_value v[], U32 argc)
{
    guru_str_append(v, v+1);
}

//================================================================
/*! (method) []
 */
__GURU__ void
c_string_slice(mrbc_value v[], U32 argc)
{
    mrbc_value *v1 = &v[1];
    mrbc_value *v2 = &v[2];

    if (argc==1 && v1->tt==GURU_TT_FIXNUM) {		// slice(n) -> String | nil
        U32 len = v->str->len;
        S32 idx = v1->i;
        S32 ch = -1;
        if (idx >= 0) {
            if (idx < len) {
                ch = *(v->str->data + idx);
            }
        }
        else {
            idx += len;
            if (idx >= 0) {
                ch = *(v->str->data + idx);
            }
        }
        if (ch < 0) goto RETURN_NIL;

        mrbc_value ret = _new(NULL, 1);
        if (!ret.str) goto RETURN_NIL;

        ret.str->data[0] = ch;
        ret.str->data[1] = '\0';

        SET_RETURN(ret);
    }
    else if (argc==2 && v1->tt==GURU_TT_FIXNUM && v2->tt==GURU_TT_FIXNUM) { 	// slice(n, len) -> String | nil
        U32 len = v->str->len;
        S32 idx = v1->i;
        if (idx < 0) idx += len;
        if (idx < 0) goto RETURN_NIL;

        S32 rlen = (v2->i < (len - idx)) ? v2->i : (len - idx);
        // min(v2->i, (len-idx))
        if (rlen < 0) goto RETURN_NIL;

        mrbc_value ret = _new((U8P)v->str->data + idx, rlen);
        if (!ret.str) goto RETURN_NIL;		// ENOMEM

        SET_RETURN(ret);
    }
    else {
    	PRINTF("Not support such case in String#[].\n");
    }
    return;

RETURN_NIL:
	SET_NIL_RETURN();
}

//================================================================
/*! (method) []=
 */
__GURU__ void
c_string_insert(mrbc_value v[], U32 argc)
{
    S32 nth;
    S32 len;
    mrbc_value *val;

    if (argc==2 &&								// self[n] = val
        v[1].tt==GURU_TT_FIXNUM &&
        v[2].tt==GURU_TT_STRING) {
        nth = v[1].i;
        len = 1;
        val = &v[2];
    }
    else if (argc==3 &&							// self[n, len] = val
             v[1].tt==GURU_TT_FIXNUM &&
             v[2].tt==GURU_TT_FIXNUM &&
             v[3].tt==GURU_TT_STRING) {
        nth = v[1].i;
        len = v[2].i;
        val = &v[3];
    }
    else {
        guru_na("case of c_string_insert");
        return;
    }

    U32 len1 = v->str->len;
    U32 len2 = val->str->len;
    if (nth < 0) nth = len1 + nth;               // adjust to positive number.
    if (len > len1 - nth) len = len1 - nth;
    if (nth < 0 || nth > len1 || len < 0) {
        PRINTF("IndexError\n");  // raise?
        return;
    }

    U8P str = (U8P)mrbc_realloc(_data(v), len1 + len2 - len + 1);
    if (!str) return;

    MEMCPY(str + nth + len2, str + nth + len, len1 - nth - len + 1);
    MEMCPY(str + nth, (U8P)_data(val), len2);
    v->str->len = len1 + len2 - len;

    v->str->data = str;

    ref_clr(v+1);
}

//================================================================
/*! (method) chomp
 */
__GURU__ void
c_string_chomp(mrbc_value v[], U32 argc)
{
    mrbc_value ret = _dup(v);
    _chomp(&ret);
    SET_RETURN(ret);
}

//================================================================
/*! (method) chomp!
 */
__GURU__ void
c_string_chomp_self(mrbc_value v[], U32 argc)
{
    if (_chomp(v)==0) {
        SET_NIL_RETURN();
    }
}

//================================================================
/*! (method) dup
 */
__GURU__ void
c_string_dup(mrbc_value v[], U32 argc)
{
    SET_RETURN(_dup(v));
}

//================================================================
/*! (method) index
 */
__GURU__ void
c_string_index(mrbc_value v[], U32 argc)
{
    S32 index;
    S32 offset;

    if (argc==1) {
        offset = 0;
    }
    else if (argc==2 && v[2].tt==GURU_TT_FIXNUM) {
        offset = v[2].i;
        if (offset < 0) offset += _size(v);
        if (offset < 0) goto NIL_RETURN;
    }
    else {
        goto NIL_RETURN;	// raise? ArgumentError
    }

    index = _index(v, v+1, offset);
    if (index < 0) goto NIL_RETURN;

    ref_clr(v+1);
    SET_INT_RETURN(index);
    return;

NIL_RETURN:
	ref_clr(v+1);
    SET_NIL_RETURN();
}

//================================================================
/*! (method) inspect
 */
#define BUF_SIZE 80

__GURU__ void
c_string_inspect(mrbc_value v[], U32 argc)
{
	const char    *hex = "0123456789ABCDEF";
    mrbc_value    ret  = guru_str_new("\"");

    U8 buf[BUF_SIZE];
    U8P p = buf;
    U8P s = (U8P)_data(v);

    for (U32 i=0; i < _size(v); i++, s++) {
        if (*s >= ' ' && *s < 0x80) {
        	*p++ = *s;
        }
        else {							// tiny isprint()
        	*p++ = '\\';
        	*p++ = 'x';
            *p++ = hex[*s >> 4];
            *p++ = hex[*s & 0x0f];
        }
    	if ((p-buf) > BUF_SIZE-5) {			// flush buffer
    		*p = '\0';
    		guru_str_append_cstr(&ret, buf);
    		p = buf;
    	}
    }
    *p++ = '\"';
    *p   = '\0';
    guru_str_append_cstr(&ret, buf);

    SET_RETURN(ret);
}

//================================================================
/*! (method) ord
 */
__GURU__ void
c_string_ord(mrbc_value v[], U32 argc)
{
    SET_INT_RETURN(_data(v)[0]);
}

//================================================================
/*! (method) split
 */
__GURU__ void
c_string_split(mrbc_value v[], U32 argc)
{
    guru_na("string#split");
}

//================================================================
/*! (method) sprintf
 */
__GURU__ void
c_object_sprintf(mrbc_value v[], U32 argc)
{
	guru_na("string#sprintf");
}

//================================================================
/*! (method) printf
 */
__GURU__ void
c_object_printf(mrbc_value v[], U32 argc)
{
	guru_na("string#printf");
}

//================================================================
/*! (method) lstrip
 */
__GURU__ void
c_string_lstrip(mrbc_value v[], U32 argc)
{
    mrbc_value ret = _dup(v);

    _strip(&ret, 0x01);	// 1: left side only

    SET_RETURN(ret);
}

//================================================================
/*! (method) lstrip!
 */
__GURU__ void
c_string_lstrip_self(mrbc_value v[], U32 argc)
{
    if (_strip(v, 0x01)==0) {	// 1: left side only
        SET_RETURN(GURU_NIL_NEW());
    }
}

//================================================================
/*! (method) rstrip
 */
__GURU__ void
c_string_rstrip(mrbc_value v[], U32 argc)
{
    mrbc_value ret = _dup(v);

    _strip(&ret, 0x02);							// 2: right side only

    SET_RETURN(ret);
}

//================================================================
/*! (method) rstrip!
 */
__GURU__ void
c_string_rstrip_self(mrbc_value v[], U32 argc)
{
    if (_strip(v, 0x02)==0) {				// 2: right side only
        SET_RETURN(GURU_NIL_NEW());			// keep refc
    }
}

//================================================================
/*! (method) strip
 */
__GURU__ void
c_string_strip(mrbc_value v[], U32 argc)
{
    mrbc_value ret = _dup(v);
    _strip(&ret, 0x03);	// 3: left and right
    SET_RETURN(ret);
}

//================================================================
/*! (method) strip!
 */
__GURU__ void
c_string_strip_self(mrbc_value v[], U32 argc)
{
    if (_strip(v, 0x03)==0) {		// 3: left and right
        SET_RETURN(GURU_NIL_NEW());	// keep refc
    }
}

//================================================================
/*! (method) to_sym
 */
__GURU__ void
c_string_to_sym(mrbc_value v[], U32 argc)
{
    SET_RETURN(guru_sym_new(_data(v)));
}

//================================================================
/*! initialize
 */
__GURU__ void
mrbc_init_class_string()
{
    guru_class *c = guru_class_string = guru_add_class("String", guru_class_object);

    guru_add_proc(c, "+",		c_string_add);
    guru_add_proc(c, "*",		c_string_mul);
    guru_add_proc(c, "size",	c_string_size);
    guru_add_proc(c, "length",	c_string_size);
    guru_add_proc(c, "to_i",	c_string_to_i);
    guru_add_proc(c, "<<",		c_string_append);
    guru_add_proc(c, "[]",		c_string_slice);
    guru_add_proc(c, "[]=",		c_string_insert);
    guru_add_proc(c, "chomp",	c_string_chomp);
    guru_add_proc(c, "chomp!",	c_string_chomp_self);
    guru_add_proc(c, "dup",		c_string_dup);
    guru_add_proc(c, "index",	c_string_index);
    guru_add_proc(c, "inspect",	c_string_inspect);
    guru_add_proc(c, "ord",		c_string_ord);
    guru_add_proc(c, "split",	c_string_split);
    guru_add_proc(c, "lstrip",	c_string_lstrip);
    guru_add_proc(c, "lstrip!",	c_string_lstrip_self);
    guru_add_proc(c, "rstrip",	c_string_rstrip);
    guru_add_proc(c, "rstrip!",	c_string_rstrip_self);
    guru_add_proc(c, "strip",	c_string_strip);
    guru_add_proc(c, "strip!",	c_string_strip_self);
    guru_add_proc(c, "to_sym",	c_string_to_sym);
    guru_add_proc(c, "intern",	c_string_to_sym);
#if GURU_USE_FLOAT
    guru_add_proc(c, "to_f",	c_string_to_f);
#endif

    guru_add_proc(guru_class_object, "sprintf",	c_object_sprintf);
    guru_add_proc(guru_class_object, "printf",	c_object_printf);
}

#endif // GURU_USE_STRING
